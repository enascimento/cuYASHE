#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <iostream>
#include <stdio.h>
#include <assert.h>
#include <hipfft/hipfft.h>
#include <unistd.h>
#include <stdlib.h>

#define PRIMES_BUCKET

hipfftHandle plan;

typedef double2 Complex;

int bitCount(uint64_t n) {

    int counter = 0;
    while(n) {
        counter += n % 2;
        n >>= 1;
    }
    return counter;
 }

__global__ void copyIntegerToComplex(Complex *a,uint64_t *b,int size){
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid < size ){
      a[tid].x =   __ull2double_rn(b[tid]);
      // printf("%ld => %f\n\n",b[tid],a[tid].x);
      a[tid].y = 0;
  }else{
    a[tid].x = 0;
    a[tid].y = 0;
  }
}

__global__ void copyAndNormalizeComplexRealPartToInteger(uint64_t *b,const Complex *a,const int size,const double scale){
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  uint64_t value;
  double fvalue;
  // double frac;
  if(tid < size ){
      fvalue = a[tid].x * scale;
      value = rint(fvalue);

      b[tid] = value;
  }
}

int check_errors(uint64_t *a, uint64_t *b, int size){
	int errors = 0;
	for(int i = 0; i < size;i++)
		if(a[i] != b[i])
			errors++;
	return errors;
}

void start_report(){
	// std::cout << "from polynomial import Polynomial" << std::endl;
	// std::cout << "import json" << std::endl;
	// std::cout << "errors = {}" << std::endl;
	// std::cout << "a = Polynomial()" << std::endl;
	// std::cout << "retorno = Polynomial()" << std::endl;
 //    std::cout << "for CRT_primesize in range(9,63):" << std::endl;
 //    std::cout << "\terrors[CRT_primesize] = {}" << std::endl;
 //    std::cout << "\tfor i in range(1,14):" << std::endl;
 //    std::cout << "\t\terrors[CRT_primesize][2**i] = 0" << std::endl;
    // std::cout << "[" << std::endl;
}

void finish_report(){
	// std::cout << "print json.dumps(errors,indent=4)" <<std::endl;
    // std::cout << "]" << std::endl;
}

void report(int SIZE,int prime_size, uint64_t *a, uint64_t *b){
	// std::cout << "CRT_primesize = " << prime_size << std::endl;
	
	// std::cout << "a.coef = [" << std::endl;
	// for(int i = 0; i < SIZE; i++)
	// 	std::cout << a[i] << ",";
	// std::cout << "]"<< std::endl;

	// std::cout << "b = a*a" << std::endl;

	// std::cout << "retorno.coef = [" << std::endl;
	// for(int i = 0; i < SIZE; i++)
	// 	std::cout << b[i] << ",";
	// std::cout << "]" << std::endl;	

 //    std::cout << "if b != retorno:" <<std::endl;
 //    std::cout << "\tfor i,coef in enumerate(b):" << std::endl;
 //    std::cout << "\t\tif i >= len(retorno.coef) or coef != retorno.coef[i]:" << std::endl;
 //    std::cout << "\t\t\terrors[CRT_primesize][" << SIZE << "] = errors[CRT_primesize][" << SIZE << "] + 1" << std::endl;

	// CRT_Primesize
	// Degree
	// A
	// B
	// C

	std::cout << prime_size << std::endl;
	std::cout << SIZE << std::endl;

	for(int i = 0; i < SIZE; i++)
		std::cout << a[i] << ",";
	std::cout << " 0" << std::endl;

	for(int i = 0; i < SIZE; i++)
		std::cout << a[i] << ",";
	std::cout << " 0" << std::endl;

	for(int i = 0; i < SIZE; i++)
		std::cout << b[i] << ",";
	std::cout << " 0" << std::endl;
}

__device__ inline Complex ComplexMul(Complex a, Complex b)
{
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}


// Complex pointwise multiplication
__global__ void mul(const Complex *a, const Complex *b,Complex *c,int size){
    const int tid = threadIdx.x + blockDim.x*blockIdx.x;

    if(tid < size  ){
        c[tid] = ComplexMul(a[tid], b[tid]);
    }else{
      c[tid].x = 0;
      c[tid].y = 0;
    }
}
/**
 * Main function
 * @param  argc [description]
 * @param  argv 1: Number of coefficients. 2-3: Interval of #Bits of each coefficient.
 * @return      [description]
 */
int main(int argc, char* argv[]){

	assert(argc >= 3);

	start_report();

	int START_RANGE = atoi(argv[1]);
	int END_RANGE = atoi(argv[2]);
	int START_COEFFBITS = atoi(argv[3]);
	int END_COEFFBITS = atoi(argv[4]);
	/////////////////
	// Setup cuFFT //
	/////////////////
	hipfftResult fftResult;

	// std::cout << "Plan created with signal size " << SIZE << std::endl;

	///////////////////////////
	// Generate polynomials. //
	///////////////////////////
	hipError_t result;

	/** Memory alloc */
	uint64_t *h_input_array;
	uint64_t *d_input_array;
	uint64_t *h_output_array;
	uint64_t *d_output_array;
	Complex *d_complex_array;

	// Degrees
    for(int SIZE = START_RANGE; SIZE < END_RANGE;SIZE *= 2){
		
		fftResult = hipfftPlan1d(&plan, SIZE, HIPFFT_Z2Z, 1);
		assert(fftResult == HIPFFT_SUCCESS);

		h_input_array = (uint64_t*)malloc(SIZE*sizeof(uint64_t));
		h_output_array = (uint64_t*)malloc(SIZE*sizeof(uint64_t));
		result = hipMalloc((void**)&d_input_array,SIZE*sizeof(uint64_t));
		assert(result == hipSuccess);
		result = hipMalloc((void**)&d_output_array,SIZE*sizeof(uint64_t));
		assert(result == hipSuccess);
		result = hipMalloc((void**)&d_complex_array,SIZE*sizeof(Complex));
		assert(result == hipSuccess);

	    result = hipMemset(d_input_array,0,SIZE*sizeof(uint64_t));
	    assert(result == hipSuccess);
	    result = hipMemset(d_output_array,0,SIZE*sizeof(uint64_t));
	    assert(result == hipSuccess);
	    result = hipMemset(d_complex_array,0,SIZE*sizeof(Complex));
	    assert(result == hipSuccess);
	    
	    // Primes
	    for(int COEFFBITS = START_COEFFBITS; COEFFBITS <= END_COEFFBITS; COEFFBITS++){

			/** Generate random coeficients mod COEFFBITS*/
			for(int j = 0; j < SIZE; j++)
				if(j < SIZE/2){
					int min = ((uint64_t)2<<(COEFFBITS-2));
					int max = ((uint64_t)2<<COEFFBITS-1); 	

					int random = (rand()%(max-min));
					h_input_array[j] = min + random;
				}
				else
					h_input_array[j] = 0;
			/** Copy */
			result = hipMemcpy(d_input_array,h_input_array,SIZE*sizeof(uint64_t),hipMemcpyHostToDevice);
			assert(result == hipSuccess);

			/** Convert to Complex */
			dim3 blockDim(32);
			dim3 gridDim((SIZE/32) + (SIZE%32));
			copyIntegerToComplex<<<gridDim,blockDim>>>(d_complex_array,d_input_array,SIZE);
			assert(hipGetLastError() == hipSuccess);

			/////////
			// FFT //
			/////////
		    fftResult = hipfftExecZ2Z(plan, (hipfftDoubleComplex *)(d_complex_array), (hipfftDoubleComplex *)(d_complex_array), HIPFFT_FORWARD);
		    assert(fftResult == HIPFFT_SUCCESS);

		    //////////
		    // Mul  //
		    //////////
			mul<<< gridDim,blockDim>>>(d_complex_array, d_complex_array,d_complex_array,SIZE);

			//////////
			// IFFT //
			//////////
		    fftResult = hipfftExecZ2Z(plan, (hipfftDoubleComplex *)(d_complex_array), (hipfftDoubleComplex *)(d_complex_array), HIPFFT_BACKWARD);
		    assert(fftResult == HIPFFT_SUCCESS);

		    ////////////
		    // Scale  //
		    ////////////
		    copyAndNormalizeComplexRealPartToInteger<<< gridDim,blockDim>>>(d_output_array,d_complex_array,SIZE,1.0f/SIZE);
			assert(hipGetLastError() == hipSuccess);

			/** Copy */
			result = hipMemcpy(h_output_array,d_output_array,SIZE*sizeof(uint64_t),hipMemcpyDeviceToHost);
			assert(result == hipSuccess);

			report(SIZE,COEFFBITS,h_input_array,h_output_array);
		}
	}
	finish_report();

	return 0;
}
