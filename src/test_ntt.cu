#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <unistd.h>
#include <assert.h>
#include <iomanip>
#include "common.h"
#include "cuda_functions.h"

#define BILLION  1000000000L
#define MILLION  1000000L
#define N 8192
#define NPOLYS 1

double compute_time_ms(struct timespec start,struct timespec stop){
  return (( stop.tv_sec - start.tv_sec )*BILLION + ( stop.tv_nsec - start.tv_nsec ))/MILLION;
}


int main(void){

  const long wN = 3;// Hard coded
  const long q = 17;

	dim3 blockDim(ADDBLOCKXDIM);
	dim3 gridDim((N*NPOLYS)/ADDBLOCKXDIM+1);

	long *h_a;
	long *d_a;
	long *h_b;
  long *d_b;
  long *h_W;
  long *d_W;

	// Alloc memory
	h_a = (long*)malloc(N*NPOLYS*sizeof(long));
	h_b = (long*)malloc(N*NPOLYS*sizeof(long));
  hipError_t result = hipMalloc((void**)&d_a,N*NPOLYS*sizeof(long));
	assert(result == hipSuccess);
  result = hipMalloc((void**)&d_b,N*NPOLYS*sizeof(long));
	assert(result == hipSuccess);

  h_W = (long*)malloc(N*N*sizeof(long));
  result = hipMalloc((void**)&d_W,N*N*sizeof(long));
	assert(result == hipSuccess);

  // Computes W
  for(int i = 0; i < N; i++)
    for(int j = 0; j < N; j++)
        // h_W[i+j*N] = (( j == 0)? 1:(h_W[i-1+j*N]*pow(wN,i)%q));
        h_W[i+j*N] = long(pow(wN,i))%q;
  std::cout << "W computed." << std::endl;
	// Generates random values
  for(int j = 0; j < NPOLYS;j++)
  	for(int i = 0; i < N/2; i++)
  		h_a[i+j*NPOLYS] = rand() % 1024;

	// Copy to GPU
  result = hipMemcpy(d_a,h_a , N*NPOLYS*sizeof(long), hipMemcpyHostToDevice);
	assert(result == hipSuccess);

  result = hipMemset((void*)d_b,0,N*NPOLYS*sizeof(long));

  result = hipMemcpy(d_W,h_W , N*N*sizeof(long), hipMemcpyHostToDevice);
	assert(result == hipSuccess);

	// Applies NTT
  host_NTT(gridDim,blockDim,d_W,d_a,d_b,N,NPOLYS);
  assert(hipGetLastError() == hipSuccess);

	// Verify if the values were really shuffled
  result = hipMemcpy(h_b,d_b,  N*NPOLYS*sizeof(long), hipMemcpyDeviceToHost);
	assert(result == hipSuccess);

	//

	hipFree(d_a);
	free(h_a);
	free(h_b);
  	std::cout << "Done." << std::endl;
}
