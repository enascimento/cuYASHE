#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <unistd.h>
#include <assert.h>
#include <iomanip>
#include <NTL/ZZ.h>

#include "common.h"
#include "cuda_functions.h"

NTL_CLIENT

#define BILLION  1000000000L
#define MILLION  1000000L
#define N 32
#define NPOLYS 1

double compute_time_ms(struct timespec start,struct timespec stop){
  return (( stop.tv_sec - start.tv_sec )*BILLION + ( stop.tv_nsec - start.tv_nsec ))/MILLION;
}


int main(void){

  const long wN = 8;// Hard coded
  const long q = 97;

	dim3 blockDim(ADDBLOCKXDIM);
	dim3 gridDim((N*NPOLYS)/ADDBLOCKXDIM+1);

	long *h_a;
	long *d_a;
	long *h_b;
  long *d_b;
  long *h_W;
  long *d_W;
  long *h_WInv;
  long *d_WInv;

	// Alloc memory
	h_a = (long*)malloc(N*NPOLYS*sizeof(long));
	h_b = (long*)malloc(N*NPOLYS*sizeof(long));
  hipError_t result = hipMalloc((void**)&d_a,N*NPOLYS*sizeof(long));
	assert(result == hipSuccess);
  result = hipMalloc((void**)&d_b,N*NPOLYS*sizeof(long));
	assert(result == hipSuccess);

  h_W = (long*)malloc(N*N*sizeof(long));
  result = hipMalloc((void**)&d_W,N*N*sizeof(long));
	assert(result == hipSuccess);
  h_WInv = (long*)malloc(N*N*sizeof(long));
  result = hipMalloc((void**)&d_WInv,N*N*sizeof(long));
	assert(result == hipSuccess);

  // Computes W
  for(int j = 0; j < N; j++)
    for(int i = 0; i < N; i++)
        // h_W[i+j*N] = (( j == 0)? 1:(h_W[i-1+j*N]*pow(wN,i)%q));
        h_W[i+j*N] = NTL::PowerMod(wN,j*i,q);

  for(int j = 0; j < N; j++)
    for(int i = 0; i < N; i++)
        h_WInv[i+j*N] = NTL::PowerMod(wN,-j*i,q);
  std::cout << "W computed." << std::endl;
	// Generates random values
  for(int j = 0; j < NPOLYS;j++)
  	for(int i = 0; i < N/2; i++)
      h_a[i+j*NPOLYS] = i;
  		// h_a[i+j*NPOLYS] = rand() % q;

  std::cout << "Input: " << std::endl;
  for(int i = 0; i < N; i++)
    std::cout << h_a[i] << std::endl;

  // std::cout << "W: " << std::endl;

  // for(int i = 0; i < N; i++)
  //   std::cout << h_W[i] << std::endl;
  // for(int i = 0; i < N; i++)
  //   std::cout << h_W[i+1*N] << std::endl;

	// Copy to GPU
  result = hipMemcpy(d_a,h_a , N*NPOLYS*sizeof(long), hipMemcpyHostToDevice);
	assert(result == hipSuccess);

  result = hipMemset((void*)d_b,0,N*NPOLYS*sizeof(long));

  result = hipMemcpy(d_W,h_W , N*N*sizeof(long), hipMemcpyHostToDevice);
	assert(result == hipSuccess);
  result = hipMemcpy(d_WInv,h_WInv , N*N*sizeof(long), hipMemcpyHostToDevice);
  assert(result == hipSuccess);

	// Applies NTT
  // Foward
  host_NTT(gridDim,blockDim,d_W,d_a,d_b,q,N,NPOLYS);
  assert(hipGetLastError() == hipSuccess);

  result = hipMemset((void*)d_a,0,N*NPOLYS*sizeof(long));

  // Inverse
  host_NTT(gridDim,blockDim,d_WInv,d_b,d_a,q,N,NPOLYS);
  assert(hipGetLastError() == hipSuccess);

	// Verify if the values were really shuffled
  result = hipMemcpy(h_b,d_a,  N*NPOLYS*sizeof(long), hipMemcpyDeviceToHost);
	assert(result == hipSuccess);

	//
  std::cout << "Output: " << std::endl;
  long NInv = NTL::InvMod(N,q);
  for(int i = 0; i < N; i++)
    std::cout << h_b[i]*NInv % q << " == " << h_a[i] << std::endl;

	hipFree(d_a);
	free(h_a);
	free(h_b);
  	std::cout << "Done." << std::endl;
}
