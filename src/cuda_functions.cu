#include "hip/hip_runtime.h"
#include "cuda_functions.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include "modop.h"

uint64_t *(CUDAFunctions::d_W) = NULL;
uint64_t *(CUDAFunctions::d_WInv) = NULL;
uint64_t CUDAFunctions::wN = 0;
int CUDAFunctions::N = 0;

///////////////////////////////////////
/// Memory operations
__global__ void realignCRTResidues(int oldSpacing,int newSpacing, uint64_t *array,uint64_t *new_array,int residuesSize,int residuesQty){
  //
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int residueId = tid / residuesSize;
  const int new_array_offset = (tid % residuesSize) + residueId*newSpacing;
  const int old_array_offset = (tid % residuesSize) + residueId*oldSpacing;

  if(new_array_offset < newSpacing*residuesQty && old_array_offset < oldSpacing*residuesQty )
    new_array[new_array_offset] = array[old_array_offset];

}


__host__ uint64_t* CUDAFunctions::callRealignCRTResidues(hipStream_t stream,int oldSpacing,int newSpacing, uint64_t *array,int residuesSize,int residuesQty){
  if(oldSpacing == newSpacing)
    return NULL;
  const int size = residuesSize*residuesQty;
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  uint64_t *d_new_array;
  hipError_t result = hipMalloc((void**)&d_new_array,newSpacing*residuesQty*sizeof(uint64_t));
  #ifdef VERBOSE
  std::cout << "hipMalloc:" << hipGetErrorString(result) << " "<< newSpacing*residuesQty*sizeof(uint64_t) << " bytes" <<std::endl;
  #endif
  assert(result == hipSuccess);

  realignCRTResidues <<< gridDim,blockDim,1,stream >>> (oldSpacing,newSpacing,array,d_new_array,residuesSize,residuesQty);
  #ifdef VERBOSE
  std::cout << gridDim.x << " " << blockDim.x << std::endl;
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif
  assert(hipGetLastError() == hipSuccess);

  return d_new_array;
}

///////////////////////////////////////

///////////////////////////////////////
/// ADD
__global__ void polynomialAddSub(const int OP,const uint64_t *a,const uint64_t *b,uint64_t *c,const int size){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  uint64_t a_value;
  uint64_t b_value;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      a_value = a[tid];
      b_value = b[tid];

      if(OP == ADD)
        a_value += b_value;
      else
        a_value -= b_value;

      c[tid] = a_value;
  }
}

__host__ uint64_t* CUDAFunctions::callPolynomialAddSub(hipStream_t stream,uint64_t *a,uint64_t *b,int size,int OP){
  // This method expects that both arrays are aligned
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  uint64_t *d_new_array;
  hipError_t result = hipMalloc((void**)&d_new_array,size*sizeof(uint64_t));
  #ifdef VERBOSE
  std::cout << "hipMalloc:" << hipGetErrorString(result) << " "<< size*sizeof(uint64_t) << " bytes" <<std::endl;
  #endif
  assert(result == hipSuccess);

  // polynomialAdd <<< gridDim,blockDim, 0, stream >>> (a,b,d_new_array,size);
  polynomialAddSub <<< gridDim,blockDim >>> (OP,a,b,d_new_array,size);
  #ifdef VERBOSE
  std::cout << gridDim.x << " " << blockDim.x << std::endl;
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif

  return d_new_array;
}
///////////////////////////////////////

///////////////////////////////////////
/// MUL

__global__ void polynomialPlainMul(const uint64_t *a,const uint64_t *b,uint64_t *c,const int N,const int NPolis){
  // Each block computes one coefficient of c
  // We need 2*N blocks for each residue!
  // 2D-blocks

  // const int tidX = threadIdx.x + blockDim.x*blockIdx.x;
  // const int tidY = threadIdx.y + blockDim.y*blockIdx.y;

  // blockDim.x == blockDim.y
  // We suppose that TILEDIM divides 2*N
  const int TILEDIM = blockDim.x;
  const int coefficient = blockIdx.x % (2*N);
  const int residueID = blockIdx.x / (2*N);
  const int residueOffset = residueID*(2*N);
  __shared__ uint64_t value;
  value = 0;

  // blockDim.x == blockDim.y
  // if(tidX < N && tidY < N){
    for(int tileY = 0;tileY < N/TILEDIM; tileY++)
      for(int tileX = 0;tileX < N/TILEDIM; tileX++){
        //      (           coefficient    ) + residue
        int i = (threadIdx.x + tileX*TILEDIM);
        int j = (threadIdx.y + tileY*TILEDIM);

        if(i + j == coefficient)
          atomicAdd((unsigned long long int *)(&value),(unsigned long long int)(a[i+residueOffset]*b[j+residueOffset]));
      }
    __syncthreads();

    if(threadIdx.x == 0 && threadIdx.y == 0)
      c[coefficient+residueOffset] = value;
      // There are 2N threads in Y axis computing this coefficient
      // atomicAdd((unsigned uint64_t uint64_t int*)(&(c[coefficient+residueOffset])),(unsigned uint64_t uint64_t int)(value));

  // }
}

__device__ uint64_t mulmod(uint64_t a, uint64_t b, uint64_t m) {
    uint64_t res = 0;
    uint64_t temp_b;

    while (a != 0) {
        if (a & 1) {
            /* Add b to res, modulo m, without overflow */
            if (b >= m - res) /* Equiv to if (res + b >= m), without overflow */
                res -= m;
            res += b;
        }
        a >>= 1;

        /* Double b, modulo m */
        temp_b = b;
        if (b >= m - b)       /* Equiv to if (2 * b >= m), without overflow */
            temp_b -= m;
        b += temp_b;
    __syncthreads();
    }
    return res;
}

typedef struct {
  unsigned long long int lo;
  unsigned long long int hi;
} my_uint128;

__device__ my_uint128 add_uint128 (my_uint128 a, my_uint128 b)
{
  my_uint128 res;
  res.lo = a.lo + b.lo;
  res.hi = a.hi + b.hi + (res.lo < a.lo);
  return res;
} 

__device__ my_uint128 add_uint64_128 (uint64_t a, my_uint128 b)
{
  my_uint128 res = {a+b.lo,b.hi}; // FALHA SE A+B.LO > 64 BITS

  // my_uint128 res;
  // res.lo = a + b.lo;
  // res.hi = b.hi + (res.lo < a);
  return res;
} 

__device__ my_uint128 sub_uint128_64 (my_uint128 a, uint64_t b)
{
  // a - b

  // my_uint128 res;
  // res.lo = a + b.lo;
  // res.hi = b.hi + (res.lo < a);
  
  my_uint128 res;
  const u_int64_t borrow = b > a.lo;

  res.lo = a.lo - b;
  res.hi = a.hi - borrow;

  return res;
} 


__device__ my_uint128 add_uint64_64 (uint64_t a, uint64_t b)
{

  my_uint128 res;
  res.lo = a + b;
  res.hi = (res.lo < a);

  return res;
} 

__device__ my_uint128 mul_uint64_128 (uint64_t a, uint64_t b)
{
  my_uint128 res = {a*b,__umul64hi(a,b)};
  return res;
} 





__device__ uint64_t s_rem (uint64_t a)
{
  // Special reduction for prime 2^64-2^32+1
  //
  // x3 * 2^96 + x2 * 2^64 + x1 * 2^32 + x0 \equiv
  // (x1+x2) * 2^32 + x0 - x3 - x2 mod p
  //
  // Here: x3 = 0, x2 = 0, x1 = (a >> 32), x0 = a-(x1 << 32)
  // const uint64_t p = 0xffffffff00000001;
  // uint64_t x3 = 0;
  // uint64_t x2 = 0;
  uint64_t x1 = (a >> 32); // Max 32 bits
  uint64_t x0 = (a & UINT32_MAX); // Max 32 bits

  // uint64_t res = ((x1+x2)<<32 + x0-x3-x2);
  uint64_t res = ((x1<<32) + x0); // Max 64 bits

  return res;
} 

__device__ uint64_t s_rem (my_uint128 a)
{
  // Special reduction for prime 2^64-2^32+1
  //
  // x3 * 2^96 + x2 * 2^64 + x1 * 2^32 + x0 \equiv
  // (x1+x2) * 2^32 + x0 - x3 - x2 mod p
  //
  // Here: x3 = 0, x2 = a.hi, x1 = (a.lo >> 32), x0 = a.lo-(x1 << 32)
  // const uint64_t p = 0xffffffff00000001;
  uint64_t x3 = (a.hi >> 32); // Max 32 bits
  uint64_t x2 = (a.hi & UINT32_MAX); // Max 32 bits
  uint64_t x1 = (a.lo >> 32); // Max 32 bits
  uint64_t x0 = (a.lo & UINT32_MAX); // Max 32 bits

  // my_uint128 x1Px232StL = {x0,(x1+x2)}; // x1 plus x2 32 Shift to Left

  // uint64_t res = sub_uint128_64(sub_uint128_64(x1Px232StL,x3),x2).lo; // -x3

  // uint64_t res = (x1 << 32);
  // res -= (x3+x2);
  // res += (x2 << 32)+x0;

  uint64_t res = ((x1+x2)<<32)+x0-x2-x3;
  return res;
} 

__global__ void NTT64(uint64_t *W,uint64_t *WInv,uint64_t *a, uint64_t *a_hat, const int N,const int NPolis,const int type){
  // This algorithm supposes that N is power of 2, divisible by 32
  // Input:
  // w: Matrix of wNs
  // a: residues
  // a_hat: output
  // N: # of coefficients of each polynomial
  // NPolis: # of residues
   const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  const int residueid = tid / (N);
  const int roffset = residueid*N;
  const int cid = tid & (N-1); // Coefficient id
  // const uint64_t P = 0xffffffff00000001;


  if(tid < N*NPolis){
    // my_uint128 value = {0,0};
    uint64_t value = 0;
    // In each iteration, computes a_hat[i]
    for(int i = 0; i < N; i++){
      uint64_t W64;
      if(type == FORWARD)
        W64 = W[i + cid*N];
      else
        W64 = WInv[i + cid*N];    

      uint64_t a64 = a[i + roffset];
      // value = (add_uint128(value, mul_uint64_128(W64,a64)));
      value = (mod_add(value, mod_mul(W64,a64)));

    }
    if(type == FORWARD)
      a_hat[cid+roffset] = (value);
    else
      a_hat[cid+roffset] = (value)/N;

  }

}



__global__ void DOUBLENTT64( uint64_t *W, uint64_t *WInv,uint64_t *a, uint64_t *a_hat,uint64_t *b, uint64_t *b_hat, const int N,const int NPolis,const int type){
  // This algorithm supposes that N is power of 2, divisible by 32
  // Input:
  // w: Matrix of wNs
  // a: residues
  // a_hat: output
  // N: # of coefficients of each polynomial
  // NPolis: # of residues
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  const int residueid = tid / N;
  const int roffset = residueid*N;
  const int cid = tid & (N-1); // Coefficient id
  // const uint64_t P = 0xffffffff00000001;


  if(tid < N*NPolis){
    uint64_t Avalue = 0;
    uint64_t Bvalue = 0;
    // my_uint128 Avalue = {0,0};
    // my_uint128 Bvalue = {0,0};
    // In each iteration, computes a_hat[i]
    for(int i = 0; i < N; i++){
      uint64_t W64;
      if(type == FORWARD)
        W64 = W[i + cid*N];
      else
        W64 = WInv[i + cid*N];

      uint64_t a64 = a[i + roffset];
      uint64_t b64 = b[i + roffset];
      // Avalue = (add_uint128(Avalue, mul_uint64_128(W64,a64)));      
      // Bvalue = (add_uint128(Bvalue, mul_uint64_128(W64,b64)));
      Avalue = (mod_add(Avalue, mod_mul(W64,a64)));      
      Bvalue = (mod_add(Bvalue, mod_mul(W64,b64)));
    }
    if(type == FORWARD){
      a_hat[cid+ roffset] = (Avalue);
      b_hat[cid+ roffset] = (Bvalue);
    }else{
      a_hat[cid+ roffset] = (Avalue)/N;
      b_hat[cid+ roffset] = (Bvalue)/N;
    }
  }
}

__global__ void polynomialNTTMul(const uint64_t *a,const uint64_t *b,uint64_t *c,const int size){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  // const uint64_t P = 0xffffffff00000001;
  uint64_t a_value;
  uint64_t b_value;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      a_value = a[tid];
      b_value = b[tid];

      // a_value = s_rem(mul_uint64_128(a_value,b_value));
      a_value = mod_mul(a_value,b_value);

      c[tid] = a_value;
  }
}

__host__ uint64_t* CUDAFunctions::callPolynomialMul(hipStream_t stream,uint64_t *a,uint64_t *b,int N,int NPolis){
  // This method expects that both arrays are aligned

  // Input:
  // stream: cudaStream
  // a: first operand
  // b: second operand
  // N: number of coefficients for each operand
  // NPolis: number of residues
  // All representations should be concatenated aligned
  assert((N>0)&&((N & (N - 1)) == 0));//Check if N is power of 2
  assert(N == CUDAFunctions::N);
  uint64_t *d_result;

  #ifdef PLAINMUL
    #ifdef VERBOSE
        std::cout << "Plain multiplication" << std::endl;
    #endif
    hipError_t result = hipMalloc((void**)&d_result,(N)*NPolis*sizeof(uint64_t));
    assert(result == hipSuccess);
    result = hipMemset((void*)d_result,0,(N)*NPolis*sizeof(uint64_t));
    assert(result == hipSuccess);

    dim3 blockDim(ADDBLOCKXDIM,ADDBLOCKXDIM);
    // int blocks = ((2*N*NPolis) % ADDBLOCKXDIM == 0? (2*N*NPolis)/ADDBLOCKXDIM : (2*N*NPolis)/ADDBLOCKXDIM+1);
    // dim3 gridDim(blocks,blocks);
    dim3 gridDim(N*NPolis,1);
    polynomialPlainMul<<<gridDim,blockDim,1,stream>>>(a,b,d_result,N,NPolis);
    assert(hipGetLastError() == hipSuccess);
  #elif defined(NTTMUL)
        // std::cout << "NTT multiplication" << std::endl;

    // Allocates memory for temporary arrays on device
    // Each polynomial's degree gets doubled
    const int size = N*NPolis;
    uint64_t *d_a;
    uint64_t *d_b;
    uint64_t *d_c;
    hipError_t result = hipMalloc((void**)&d_a,size*sizeof(uint64_t));
    assert(result == hipSuccess);
    // result = hipMemset((void*)d_a,0,size*sizeof(uint64_t));
    // assert(result == hipSuccess);
    result = hipMalloc((void**)&d_b,size*sizeof(uint64_t));
    assert(result == hipSuccess);
    // result = hipMemset((void*)d_b,0,size*sizeof(uint64_t));
    // assert(result == hipSuccess);    
    result = hipMalloc((void**)&d_c,size*sizeof(uint64_t));
    assert(result == hipSuccess);
    result = hipMalloc((void**)&d_result,size*sizeof(uint64_t));
    assert(result == hipSuccess);
    
    dim3 blockDim(ADDBLOCKXDIM);
    dim3 gridDim((size)/ADDBLOCKXDIM); // We expect that ADDBLOCKXDIM always divice size

    // Forward 
    DOUBLENTT64<<<gridDim,blockDim,1,stream>>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,a,d_a,b,d_b,N,NPolis,FORWARD);
    assert(hipGetLastError() == hipSuccess);

    // Multiply
    polynomialNTTMul<<<gridDim,blockDim,1,stream>>>(d_a,d_b,d_c,N*NPolis);

    // Inverse    
    NTT64<<<gridDim,blockDim,1,stream>>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,d_c,d_result,N,NPolis,INVERSE);
    result = hipDeviceSynchronize();
    assert(result == hipSuccess);
    assert(hipGetLastError() == hipSuccess);

    checkCudaErrors(hipPeekAtLastError());
    checkCudaErrors(hipDeviceSynchronize());

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
  #endif

  return d_result;
}
