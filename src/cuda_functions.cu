#include "hip/hip_runtime.h"
#include "cuda_functions.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>

uint32_t *(CUDAFunctions::d_W) = NULL;
uint32_t *(CUDAFunctions::d_WInv) = NULL;
uint64_t CUDAFunctions::wN = 0;

///////////////////////////////////////
/// Memory operations
__global__ void realignCRTResidues(int oldSpacing,int newSpacing, uint32_t *array,uint32_t *new_array,int residuesSize,int residuesQty){
  //
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int residueId = tid / residuesSize;
  const int new_array_offset = (tid % residuesSize) + residueId*newSpacing;
  const int old_array_offset = (tid % residuesSize) + residueId*oldSpacing;

  if(new_array_offset < newSpacing*residuesQty && old_array_offset < oldSpacing*residuesQty )
    new_array[new_array_offset] = array[old_array_offset];

}


__host__ uint32_t* CUDAFunctions::callRealignCRTResidues(hipStream_t stream,int oldSpacing,int newSpacing, uint32_t *array,int residuesSize,int residuesQty){
  if(oldSpacing == newSpacing)
    return NULL;
  const int size = residuesSize*residuesQty;
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  uint32_t *d_new_array;
  hipError_t result = hipMalloc((void**)&d_new_array,newSpacing*residuesQty*sizeof(uint32_t));
  #ifdef VERBOSE
  std::cout << "hipMalloc:" << hipGetErrorString(result) << " "<< newSpacing*residuesQty*sizeof(uint32_t) << " bytes" <<std::endl;
  #endif
  assert(result == hipSuccess);

  realignCRTResidues <<< gridDim,blockDim,1,stream >>> (oldSpacing,newSpacing,array,d_new_array,residuesSize,residuesQty);
  #ifdef VERBOSE
  std::cout << gridDim.x << " " << blockDim.x << std::endl;
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif
  assert(hipGetLastError() == hipSuccess);

  return d_new_array;
}

///////////////////////////////////////

///////////////////////////////////////
/// ADD
__global__ void polynomialAddSub(const int OP,const uint32_t *a,const uint32_t *b,uint32_t *c,const int size){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  uint32_t a_value;
  uint32_t b_value;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      a_value = a[tid];
      b_value = b[tid];

      if(OP == ADD)
        a_value += b_value;
      else
        a_value -= b_value;

      c[tid] = a_value;
  }
}

__host__ uint32_t* CUDAFunctions::callPolynomialAddSub(hipStream_t stream,uint32_t *a,uint32_t *b,int size,int OP){
  // This method expects that both arrays are aligned
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  uint32_t *d_new_array;
  hipError_t result = hipMalloc((void**)&d_new_array,size*sizeof(uint32_t));
  #ifdef VERBOSE
  std::cout << "hipMalloc:" << hipGetErrorString(result) << " "<< size*sizeof(uint32_t) << " bytes" <<std::endl;
  #endif
  assert(result == hipSuccess);

  // polynomialAdd <<< gridDim,blockDim, 0, stream >>> (a,b,d_new_array,size);
  polynomialAddSub <<< gridDim,blockDim >>> (OP,a,b,d_new_array,size);
  #ifdef VERBOSE
  std::cout << gridDim.x << " " << blockDim.x << std::endl;
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif

  return d_new_array;
}
///////////////////////////////////////

///////////////////////////////////////
/// MUL

__global__ void polynomialPlainMul(const uint32_t *a,const uint32_t *b,uint32_t *c,const int N,const int NPolis){
  // Each block computes one coefficient of c
  // We need 2*N blocks for each residue!
  // 2D-blocks

  // const int tidX = threadIdx.x + blockDim.x*blockIdx.x;
  // const int tidY = threadIdx.y + blockDim.y*blockIdx.y;

  // blockDim.x == blockDim.y
  // We suppose that TILEDIM divides 2*N
  const int TILEDIM = blockDim.x;
  const int coefficient = blockIdx.x % (2*N);
  const int residueID = blockIdx.x / (2*N);
  const int residueOffset = residueID*(2*N);
  __shared__ uint32_t value;
  value = 0;

  // blockDim.x == blockDim.y
  // if(tidX < N && tidY < N){
    for(int tileY = 0;tileY < N/TILEDIM; tileY++)
      for(int tileX = 0;tileX < N/TILEDIM; tileX++){
        //      (           coefficient    ) + residue
        int i = (threadIdx.x + tileX*TILEDIM);
        int j = (threadIdx.y + tileY*TILEDIM);

        if(i + j == coefficient)
          atomicAdd((&value),(a[i+residueOffset]*b[j+residueOffset]));
      }
    __syncthreads();

    if(threadIdx.x == 0 && threadIdx.y == 0)
      c[coefficient+residueOffset] = value;
      // There are 2N threads in Y axis computing this coefficient
      // atomicAdd((unsigned uint32_t uint32_t int*)(&(c[coefficient+residueOffset])),(unsigned uint32_t uint32_t int)(value));

  // }
}

__global__ void NTT32(uint32_t *W,uint32_t *WInv,uint32_t *a, uint32_t *a_hat, const int N,const int NPolis,const uint64_t P,const int type){
  // This algorithm supposes that N is power of 2, divisible by 32
  // Input:
  // w: Matrix of wNs
  // a: residues
  // a_hat: output
  // N: # of coefficients of each polynomial
  // NPolis: # of residues
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  const int residueid = tid / (N);
  const int roffset = residueid*N;
  const int cid = tid & (N-1); // Coefficient id
  // const double invk = (double)(1<<30) / P;
  uint32_t *w;
  if(type == FORWARD)
    w = W;
  else
    w = WInv;

  // const inteiro p = 0xffffffff00000001;
  if(tid < N*NPolis){
    uint64_t value = 0;
    // In each iteration, computes a_hat[i]
    for(int i = 0; i < N; i++){
      uint64_t W64 = w[i + cid*N];
      uint64_t a64 = a[i + roffset];
      value = (value + W64*a64)%P;
      // value = value + mul_m(W64,a64,P,invk);
    }
    if(type == FORWARD)
      a_hat[cid+roffset] = value % P;
    else
      a_hat[cid+roffset] = (value % P)/N;

  }

}



__global__ void DOUBLENTT32( uint32_t *W, uint32_t *WInv,uint32_t *a, uint32_t *a_hat,uint32_t *b, uint32_t *b_hat, const int N,const int NPolis,const uint64_t P,const int type){
  // This algorithm supposes that N is power of 2, divisible by 32
  // Input:
  // w: Matrix of wNs
  // a: residues
  // a_hat: output
  // N: # of coefficients of each polynomial
  // NPolis: # of residues

  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  const int residueid = tid / N;
  const int roffset = residueid*N;
  const int cid = tid & (N-1); // Coefficient id
  uint32_t *w;
  if(type == FORWARD)
    w = W;
  else
    w = WInv;

  // const uint32_t p = 0xffffffff00000001;
  if(tid < N*NPolis){
    uint64_t Avalue = 0;
    uint64_t Bvalue = 0;
    // In each iteration, computes a_hat[i]
    for(int i = 0; i < N; i++){

      uint64_t W64 = w[i + cid*N];
      uint64_t a64 = a[i + roffset];
      uint64_t b64 = b[i + roffset];
      Avalue = (Avalue + W64*a64)%P;      
      Bvalue = (Bvalue + W64*b64)%P;
    }
    if(type == FORWARD){
      a_hat[cid+ roffset] = Avalue % P;
      b_hat[cid+ roffset] = Bvalue % P;
    }else{
      a_hat[cid+ roffset] = (Avalue % P)/N;
      b_hat[cid+ roffset] = (Bvalue % P)/N;
    }
  }

}

__global__ void polynomialNTTMul(const uint32_t *a,const uint32_t *b,uint32_t *c,const int size,const uint64_t P){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  uint64_t a_value;
  uint64_t b_value;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      a_value = a[tid];
      b_value = b[tid];

      a_value = (a_value*b_value) % P;

      c[tid] = a_value;
  }
}

__host__ uint32_t* CUDAFunctions::callPolynomialMul(hipStream_t stream,uint32_t *a,uint32_t *b,int N,int NPolis){
  // This method expects that both arrays are aligned

  // Input:
  // stream: cudaStream
  // a: first operand
  // b: second operand
  // N: number of coefficients for each operand
  // NPolis: number of residues
  // All representations should be concatenated aligned
  assert((N>0)&&((N & (N - 1)) == 0));//Check if N is power of 2
  uint32_t *d_result;

  #ifdef PLAINMUL
    #ifdef VERBOSE
        std::cout << "Plain multiplication" << std::endl;
    #endif
    hipError_t result = hipMalloc((void**)&d_result,(2*N)*NPolis*sizeof(uint32_t));
    assert(result == hipSuccess);
    result = hipMemset((void*)d_result,0,(2*N)*NPolis*sizeof(uint32_t));
    assert(result == hipSuccess);

    dim3 blockDim(ADDBLOCKXDIM,ADDBLOCKXDIM);
    // int blocks = ((2*N*NPolis) % ADDBLOCKXDIM == 0? (2*N*NPolis)/ADDBLOCKXDIM : (2*N*NPolis)/ADDBLOCKXDIM+1);
    // dim3 gridDim(blocks,blocks);
    dim3 gridDim(2*N*NPolis,1);
    polynomialPlainMul<<<gridDim,blockDim,1,stream>>>(a,b,d_result,N,NPolis);
    assert(hipGetLastError() == hipSuccess);
  #elif defined(NTTMUL)
        std::cout << "NTT multiplication" << std::endl;

    // Allocates memory for temporary arrays on device
    // Each polynomial's degree gets doubled
    const int size = N*NPolis;
    uint32_t *d_a;
    uint32_t *d_b;
    uint32_t *d_c;
    hipError_t result = hipMalloc((void**)&d_a,size*sizeof(uint32_t));
    assert(result == hipSuccess);
    result = hipMemset((void*)d_a,0,size*sizeof(uint32_t));
    assert(result == hipSuccess);
    result = hipMalloc((void**)&d_b,size*sizeof(uint32_t));
    assert(result == hipSuccess);
    result = hipMemset((void*)d_b,0,size*sizeof(uint32_t));
    assert(result == hipSuccess);    
    result = hipMalloc((void**)&d_c,size*sizeof(uint32_t));
    assert(result == hipSuccess);
    result = hipMalloc((void**)&d_result,size*NPolis*sizeof(uint32_t));
    assert(result == hipSuccess);
    
    dim3 blockDim(ADDBLOCKXDIM);
    dim3 gridDim((size)/ADDBLOCKXDIM); // We expect that ADDBLOCKXDIM always divice size

    // Forward 
    DOUBLENTT32<<<gridDim,blockDim,1,stream>>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,a,d_a,b,d_b,N,NPolis,CUDAFunctions::P,FORWARD);
    assert(hipGetLastError() == hipSuccess);

    // Multiply
    polynomialNTTMul<<<gridDim,blockDim,1,stream>>>(d_a,d_b,d_c,N*NPolis,P);

    // Inverse    
    NTT32<<<gridDim,blockDim,1,stream>>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,d_c,d_result,N,NPolis,CUDAFunctions::P,INVERSE);
    assert(hipGetLastError() == hipSuccess);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
  #endif

  return d_result;
}
