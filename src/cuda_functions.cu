#include "hip/hip_runtime.h"
#include "cuda_functions.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>

///////////////////////////////////////
/// Memory operations
__global__ void realignCRTResidues(int oldSpacing,int newSpacing, long *array,long *new_array,int residuesSize,int residuesQty){
  //
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int residueId = tid / residuesSize;
  const int new_array_offset = (tid % residuesSize) + residueId*newSpacing;
  const int old_array_offset = (tid % residuesSize) + residueId*oldSpacing;

  if(new_array_offset < newSpacing*residuesQty && old_array_offset < oldSpacing*residuesQty )
    new_array[new_array_offset] = array[old_array_offset];

}


__host__ long* CUDAFunctions::callRealignCRTResidues(hipStream_t stream,int oldSpacing,int newSpacing, long *array,int residuesSize,int residuesQty){
  if(oldSpacing == newSpacing)
    return NULL;
  const int size = residuesSize*residuesQty;
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  long *d_new_array;
  hipError_t result = hipMalloc((void**)&d_new_array,newSpacing*residuesQty*sizeof(long));
  #ifdef VERBOSE
  std::cout << "hipMalloc:" << hipGetErrorString(result) << " "<< newSpacing*residuesQty*sizeof(long) << " bytes" <<std::endl;
  #endif
  assert(result == hipSuccess);

  realignCRTResidues <<< gridDim,blockDim,1,stream >>> (oldSpacing,newSpacing,array,d_new_array,residuesSize,residuesQty);
  #ifdef VERBOSE
  std::cout << gridDim.x << " " << blockDim.x << std::endl;
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif
  assert(hipGetLastError() == hipSuccess);

  return d_new_array;
}

///////////////////////////////////////

///////////////////////////////////////
/// ADD
__global__ void polynomialAddSub(const int OP,const long *a,const long *b,long *c,const int size){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  long a_value;
  long b_value;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      a_value = a[tid];
      b_value = b[tid];

      if(OP == ADD)
        a_value += b_value;
      else
        a_value -= b_value;

      c[tid] = a_value;
  }
}

__host__ long* CUDAFunctions::callPolynomialAddSub(hipStream_t stream,long *a,long *b,int size,int OP){
  // This method expects that both arrays are aligned
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  long *d_new_array;
  hipError_t result = hipMalloc((void**)&d_new_array,size*sizeof(long));
  #ifdef VERBOSE
  std::cout << "hipMalloc:" << hipGetErrorString(result) << " "<< size*sizeof(long) << " bytes" <<std::endl;
  #endif
  assert(result == hipSuccess);

  // polynomialAdd <<< gridDim,blockDim, 0, stream >>> (a,b,d_new_array,size);
  polynomialAddSub <<< gridDim,blockDim >>> (OP,a,b,d_new_array,size);
  #ifdef VERBOSE
  std::cout << gridDim.x << " " << blockDim.x << std::endl;
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif

  return d_new_array;
}
///////////////////////////////////////

///////////////////////////////////////
/// MUL

__global__ void polynomialPlainMul(const long *a,const long *b,long *c,const int N,const int NPolis){
  // Each block computes one coefficient of c
  // We need 2*N blocks for each residue!
  // 2D-blocks

  // const int tidX = threadIdx.x + blockDim.x*blockIdx.x;
  // const int tidY = threadIdx.y + blockDim.y*blockIdx.y;

  // blockDim.x == blockDim.y
  // We suppose that TILEDIM divides 2*N
  const int TILEDIM = blockDim.x;
  const int coefficient = blockIdx.x % (2*N);
  const int residueID = blockIdx.x / (2*N);
  const int residueOffset = residueID*(2*N);
  __shared__ long value;
  value = 0;

  // blockDim.x == blockDim.y
  // if(tidX < N && tidY < N){
    for(int tileY = 0;tileY < N/TILEDIM; tileY++)
      for(int tileX = 0;tileX < N/TILEDIM; tileX++){
        //      (           coefficient    ) + residue
        int i = (threadIdx.x + tileX*TILEDIM);
        int j = (threadIdx.y + tileY*TILEDIM);

        if(i + j == coefficient)
          atomicAdd((unsigned long long int*)(&value),(unsigned long long int)(a[i+residueOffset]*b[j+residueOffset]));
      }
    __syncthreads();

    if(threadIdx.x == 0 && threadIdx.y == 0)
      c[coefficient+residueOffset] = value;
      // There are 2N threads in Y axis computing this coefficient
      // atomicAdd((unsigned long long int*)(&(c[coefficient+residueOffset])),(unsigned long long int)(value));

  // }
}

__host__ long* CUDAFunctions::callPolynomialMul(hipStream_t stream,long *a,long *b,int N,int NPolis){
  // This method expects that both arrays are aligned

  // Input:
  // stream: cudaStream
  // a: first operand
  // b: second operand
  // N: number of coefficients for each operand
  // NPolis: number of residues
  // All representations should be concatenated aligned
  assert((N>0)&&((N & (N - 1)) == 0));//Check if N is power of 2
  long *d_result;

  #ifdef PLAINMUL
    #ifdef VERBOSE
        std::cout << "Plain multiplication" << std::endl;
    #endif
    hipError_t result = hipMalloc((void**)&d_result,(2*N)*NPolis*sizeof(long));
    assert(result == hipSuccess);
    result = hipMemset((void*)d_result,0,(2*N)*NPolis*sizeof(long));
    assert(result == hipSuccess);

    dim3 blockDim(ADDBLOCKXDIM,ADDBLOCKXDIM);
    // int blocks = ((2*N*NPolis) % ADDBLOCKXDIM == 0? (2*N*NPolis)/ADDBLOCKXDIM : (2*N*NPolis)/ADDBLOCKXDIM+1);
    // dim3 gridDim(blocks,blocks);
    dim3 gridDim(2*N*NPolis,1);
    polynomialPlainMul<<<gridDim,blockDim,1,stream>>>(a,b,d_result,N,NPolis);
    assert(hipGetLastError() == hipSuccess);
  #else

    // To-do
    throw "Polynomial multiplication not implemented!";
    // Allocates memory for temporary arrays on device
    // Each polynomial's degree gets doubled
    long *d_a;
    long *d_b;
    hipError_t result = hipMalloc((void**)&d_a,2*size*sizeof(long));
    assert(result == hipSuccess);
    hipError_t result = hipMalloc((void**)&d_b,2*size*sizeof(long));
    assert(result == hipSuccess);

    dim3 blockDim(32);
    dim3 gridDim((2*size)/32+1);
    NTT<<<gridDim,blockDim,1,stream>>>(a,d_a,N,size);
  #endif

  return d_result;
}
