#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include "cuda_functions.h"
#include "cuda_bn.h"
#include "settings.h"
#include "polynomial.h"


#ifdef NTTMUL
// #define PRIMEP (int)2147483647
// #define PRIMITIVE_ROOT (int)7;//2^31-1 fails the test(P-1)%N
// #define PRIMEP (uint32_t)4294955009
// #define PRIMITIVE_ROOT (int)3
#define PRIMEP (uint64_t)18446744069414584321
#define PRIMITIVE_ROOT (int)7

 ZZ PZZ = to_ZZ(PRIMEP); 

cuyasheint_t CUDAFunctions::wN = 0;
cuyasheint_t *CUDAFunctions::d_W = NULL;
cuyasheint_t *CUDAFunctions::d_WInv = NULL;
cuyasheint_t *CUDAFunctions::d_inner_results = NULL;
cuyasheint_t *CUDAFunctions::d_inner_results_used = NULL;

#elif defined(CUFFTMUL)
hipfftHandle CUDAFunctions::plan;
typedef double2 Complex;
#endif
int CUDAFunctions::N = 0;

///////////////////////
// Memory operations //
///////////////////////


__global__ void realignCRTResidues(int oldSpacing,int newSpacing, cuyasheint_t *array,cuyasheint_t *new_array,int residuesSize,int residuesQty){
  //
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int residueId = (newSpacing < oldSpacing ? tid / newSpacing: tid / oldSpacing);
  const int new_array_offset = (newSpacing < oldSpacing ? (tid % newSpacing) + residueId*newSpacing:(tid % oldSpacing) + residueId*newSpacing);
  const int old_array_offset = (newSpacing < oldSpacing ? (tid % newSpacing) + residueId*oldSpacing:(tid % oldSpacing) + residueId*oldSpacing);

  if(new_array_offset < newSpacing*residuesQty && old_array_offset < oldSpacing*residuesQty )
    new_array[new_array_offset] = array[old_array_offset];

}


__host__ cuyasheint_t* CUDAFunctions::callRealignCRTResidues(hipStream_t stream,
                                                              const int oldSpacing,
                                                              const int newSpacing,
                                                              cuyasheint_t *array,
                                                              const int residuesSize,
                                                              const int residuesQty){
  if(oldSpacing == newSpacing)
    return NULL;
  #ifdef VERBOSE
  std::cout << "Realigning..." << std::endl;
  #endif
  
  int size;
  if(newSpacing < oldSpacing)
    size = newSpacing*residuesQty;
  else
    size = oldSpacing *residuesQty;
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  cuyasheint_t *d_new_array;
  hipError_t result = hipMalloc((void**)&d_new_array,newSpacing*residuesQty*sizeof(cuyasheint_t));
  assert(result == hipSuccess);
  result = hipMemsetAsync( d_new_array,
                            0,
                            newSpacing*residuesQty*sizeof(cuyasheint_t),
                            stream);
  assert(result == hipSuccess);

  realignCRTResidues <<< gridDim,blockDim, 1, stream >>> (oldSpacing,newSpacing,array,d_new_array,residuesSize,residuesQty);
  result = hipGetLastError();
  assert(result == hipSuccess);

  return d_new_array;
}

///////////////////////////////////////
/// ADD
///////////////////////////////////////

__global__ void polynomialAddSub(const int OP,const cuyasheint_t *a,const cuyasheint_t *b,cuyasheint_t *c,const int size,const int N){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      if(OP == ADD){
        c[tid] = a[tid] + b[tid];
        if(c[tid] < a[tid])
          printf("Overflow!");
      }else
        c[tid] = a[tid] - b[tid];
  }
}

__host__ void CUDAFunctions::callPolynomialAddSub(cuyasheint_t *c,cuyasheint_t *a,cuyasheint_t *b,int size,int OP,hipStream_t stream){
  // This method expects that both arrays are aligned
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  polynomialAddSub <<< gridDim,blockDim,0,stream  >>> (OP,a,b,c,size,N);
  assert(hipGetLastError() == hipSuccess);
  #ifdef VERBOSE
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif
}

__global__ void polynomialAddSubInPlace(const int OP, cuyasheint_t *a,const cuyasheint_t *b,const int size,const int N){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  // const int rid = tid / N; // Residue id
  cuyasheint_t a_value;
  cuyasheint_t b_value;

  if(tid < size ){
      // printf("A[0]: %d\nB[0]: %d\n\n",a[tid],b[tid]);
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      a_value = a[tid];
      b_value = b[tid];

      if(OP == ADD){
        a_value += b_value;
        if(a_value < a[tid])
          printf("Overflow!\n");
      }else
        a_value -= b_value;

      a[tid] = a_value;
  }
}

__host__ void CUDAFunctions::callPolynomialAddSubInPlace(hipStream_t stream,cuyasheint_t *a,cuyasheint_t *b,int size,int OP){
  // This method expects that both arrays are aligned
  // Add and store in array a
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  polynomialAddSubInPlace <<< gridDim,blockDim,0,stream >>> (OP,a,b,size,N);
  assert(hipGetLastError() == hipSuccess);
  #ifdef VERBOSE
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif
}
///////////////////////////////////////

///////////////////////////////////////
/// MUL

#ifdef PLAINMUL
__global__ void polynomialPlainMul(const cuyasheint_t *a,const cuyasheint_t *b,cuyasheint_t *c,const int N,const int NPolis){
  // Each block computes one coefficient of c
  // We need 2*N blocks for each residue!
  // 2D-blocks

  // const int tidX = threadIdx.x + blockDim.x*blockIdx.x;
  // const int tidY = threadIdx.y + blockDim.y*blockIdx.y;

  // blockDim.x == blockDim.y
  // We suppose that TILEDIM divides 2*N
  const int TILEDIM = blockDim.x;
  const int coefficient = blockIdx.x % (2*N);
  const int residueID = blockIdx.x / (2*N);
  const int residueOffset = residueID*(2*N);
  __shared__ cuyasheint_t value;
  value = 0;

  // blockDim.x == blockDim.y
  // if(tidX < N && tidY < N){
    for(int tileY = 0;tileY < N/TILEDIM; tileY++)
      for(int tileX = 0;tileX < N/TILEDIM; tileX++){
        //      (           coefficient    ) + residue
        int i = (threadIdx.x + tileX*TILEDIM);
        int j = (threadIdx.y + tileY*TILEDIM);

        if(i + j == coefficient)
          atomicAdd((unsigned long long int *)(&value),(unsigned long long int)(a[i+residueOffset]*b[j+residueOffset]));
      }
    __syncthreads();

    if(threadIdx.x == 0 && threadIdx.y == 0)
      c[coefficient+residueOffset] = value;
      // There are 2N threads in Y axis computing this coefficient
      // atomicAdd((unsigned cuyasheint_t cuyasheint_t int*)(&(c[coefficient+residueOffset])),(unsigned cuyasheint_t cuyasheint_t int)(value));

  // }
}
#elif defined(CUFFTMUL)

__global__ void copyIntegerToComplex(Complex *a,cuyasheint_t *b,int size){
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid < size ){
      a[tid].x =   __ull2double_rn(b[tid]);
      // printf("%ld => %f\n\n",b[tid],a[tid].x);
      a[tid].y = 0;
  }else{
    a[tid].x = 0;
    a[tid].y = 0;
  }
}

__global__ void copyAndRealignIntegerToComplex(Complex *a,cuyasheint_t *b,const unsigned oldSpacing,const unsigned int newSpacing,const unsigned int residuesQty){
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int residueId = (newSpacing < oldSpacing ? tid / newSpacing: tid / oldSpacing);
  const int new_array_offset = (newSpacing < oldSpacing ? (tid % newSpacing) + residueId*newSpacing:(tid % oldSpacing) + residueId*newSpacing);
  const int old_array_offset = (newSpacing < oldSpacing ? (tid % newSpacing) + residueId*oldSpacing:(tid % oldSpacing) + residueId*oldSpacing);

  if(new_array_offset < newSpacing*residuesQty && old_array_offset < oldSpacing*residuesQty ){
      a[new_array_offset].x =  __ull2double_rn(b[old_array_offset]);
      a[new_array_offset].y = 0;
  }
}

__global__ void copyAndNormalizeComplexRealPartToInteger(cuyasheint_t *b,const Complex *a,const int size,const double scale,const int N){
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int rid = tid / N; // Residue id
  cuyasheint_t value;
  double fvalue;
  // double frac;
  if(tid < size ){
      fvalue = a[tid].x * scale;
      value = rint(fvalue);

      b[tid] = value;
  }
}
////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex multiplication
static __device__ inline Complex ComplexMul(Complex a, Complex b)
{
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}


// Complex pointwise multiplication
static __global__ void polynomialcuFFTMul(const Complex *a, const Complex *b,Complex *c,int size_c,int size){
    const int tid = threadIdx.x + blockDim.x*blockIdx.x;

    if(tid < size  ){
        c[tid] = ComplexMul(a[tid], b[tid]);
    }else{
      c[tid].x = 0;
      c[tid].y = 0;
    }
}
#elif defined(NTTMUL)

__host__ __device__ bool overflow(const uint64_t a, const uint64_t b){
  // True if a+b will result in a integer overflow.
  return (a+b) < a;
  // return lessThan((a+b),a);
}

__host__ __device__ uint64_t s_rem (uint64_t a)
{
  // Special reduction for prime 2^64-2^32+1
  //
  // x3 * 2^96 + x2 * 2^64 + x1 * 2^32 + x0 \equiv
  // (x1+x2) * 2^32 + x0 - x3 - x2 mod p
  //
  // Here: x3 = 0, x2 = 0, x1 = (a >> 32), x0 = a-(x1 << 32)
  // const uint64_t p = 0xffffffff00000001;
  // uint64_t x3 = 0;
  // uint64_t x2 = 0;

  uint64_t x1 = (a >> 32);
  uint64_t x0 = (a & UINT32_MAX);

  // uint64_t res = ((x1+x2)<<32 + x0-x3-x2);
  uint64_t res = ((x1<<32) + x0);

  if(res >= PRIMEP){
    res -= PRIMEP;
    x1 = (res >> 32);
    x0 = (res & UINT32_MAX);
    res = ((x1<<32) + x0);
  }

  return res;
}

__host__ __device__  uint64_t s_mul(uint64_t a,uint64_t b){
  // Multiply and reduce a and b by prime 2^64-2^32+1
  #ifdef __CUDA_ARCH__
  const uint64_t GAP = (UINT64_MAX-PRIMEP+1);

  const uint64_t cHi = __umul64hi(a,b);
  const uint64_t cLo = a*b;


  // Reduce
  const uint64_t x3 = (cHi >> 32);
  const uint64_t x2 = (cHi & UINT32_MAX);
  const uint64_t x1 = (cLo >> 32);
  const uint64_t x0 = (cLo & UINT32_MAX);

  const uint64_t X1 = (x1<<32);
  const uint64_t X2 = (x2<<32);

  ///////////////////////////////
  //
  // Here we can see three kinds of overflows:
  //
  // * Negative overflow: Result is negative. 
  // Since uint64_t uses mod UINT64_MAX, we need to translate to the correct value mod PRIMEP.
  // * Simple overflow: Result is bigger than PRIMEP but not enough to exceed UINT64_MAX.
  //  We solve this in the same way we solve negative overflow, just translate to the correct value mod PRIMEP.
  // * Double overflow

  uint64_t res = X1+X2+x0-x2-x3;
  const bool testA = (x2+x3 > X1+X2+x0) && !( overflow(X1,X2) ||  overflow(X1+X2,x0) ); // Negative overflow
  const bool testB = ( res >= PRIMEP ); // Simple overflow
  const bool testC = (overflow(X1,X2) || overflow(X1+X2,x0)) && (X1+X2+x0 > x2+x3); // Double overflow

  // This avoids conditional branchs
  // res = (PRIMEP-res)*(testA) + (res-PRIMEP)*(!testA && testB) + (PRIMEP - (UINT64_MAX-res))*(!testA && !testB && testC) + (res)*(!testA && !testB && !testC);
  res =   (PRIMEP-res)*(testA) 
        + (res-PRIMEP)*(!testA && testB) 
        + (res+GAP)*(!testA && !testB && testC) 
        + (res)*(!testA && !testB && !testC);

   #else
  uint64_t res = (((__uint128_t)a) * ((__uint128_t)b) )%PRIMEP;
  #endif
  return res;
}
__host__ __device__  uint64_t s_add(uint64_t a,uint64_t b){
  // Add and reduce a and b by prime 2^64-2^32+1
  // 4294967295L == UINT64_MAX - P
  uint64_t res = a+b;
  res += (res < a)*4294967295L;

  return s_rem(res);
}


__host__ __device__ uint64_t s_sub(uint64_t a,uint64_t b){
  // Computes a-b % P
  // 4294967295L == UINT64_MAX - P

  uint64_t res;
  // if(b > a){
  //   res = PRIMEP;
  //   res -= b;
  //   res += a;
  // }
  // else
  //   res = a-b;
  res = (a-b) + (b > a)*PRIMEP; 

  #ifdef __CUDA_ARCH__
  __syncthreads();
  #endif
  return res;
}

__host__ __device__ void butterfly(uint64_t *v){
  // Butterfly
  const uint64_t v0 = s_rem(v[0]);
  const uint64_t v1 = s_rem(v[1]);
  v[0] = s_add(v0,v1);
  v[1] = s_sub(v0,v1);
}

__host__ __device__ int expand(int idxL, int N1, int N2){
	return (idxL/N1)*N1*N2 + (idxL%N1);
}

__global__ void NTTScale(cuyasheint_t *data,const int size,const int N){
  const unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
  // const unsigned int cid = (tid/N)*N + (tid%N); // residueId*resideSize + coefficient 
  if( tid < size )
    data[tid] /= N;
} 

__host__ __device__ void NTTIteration(cuyasheint_t *W,
                                      cuyasheint_t *WInv,
                                      const int residue_index,
                                      const int j,
                                      const int N,
                                      const int R,
                                      const int Ns,
                                      const cuyasheint_t* data0,
                                      cuyasheint_t *data1,
                                      const int type){
	uint64_t v[2] = {0,0};
	const int idxS = j+residue_index;
  int w_index = ((j%Ns)*N)/(Ns*R);

  for(int r=0; r<R; r++)
    if(type == FORWARD)
      v[r] = s_mul(data0[idxS+r*N/R],W[w_index*r]);
    else
      v[r] = s_mul(data0[idxS+r*N/R],WInv[w_index*r]);

	butterfly(v);

	const int idxD = expand(j,Ns,R)+residue_index;
	for(int r=0; r<R;r++){
  		data1[idxD+r*Ns] = v[r];
    #ifdef __CUDA_ARCH__
    __syncthreads();
    #endif
  }
}

__global__ void NTT(cuyasheint_t *d_W,cuyasheint_t *d_WInv,const int N, const int R, const int Ns, cuyasheint_t* dataI, cuyasheint_t* dataO,const int type){

  const int residue_index = (blockIdx.x)*N;
  for(int i = 0; i < N/R; i += 1024){
    // " Threads virtuais "
    const int j = (threadIdx.x+i);
    if( j < N)
      NTTIteration(d_W,d_WInv,residue_index,j, N, R, Ns, dataI, dataO,type);
    __syncthreads();
  }
}

__host__ void CUDAFunctions::callNTT(const int N, const int NPolis,cuyasheint_t* dataI, cuyasheint_t* dataO,const int type){

  const int RADIX = 2;
  dim3 blockDim(std::min(N/RADIX,1024));
  dim3 gridDim(NPolis);

  for(int Ns=1; Ns<N; Ns*=RADIX){
    NTT<<<gridDim,blockDim>>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,RADIX,Ns,dataI,dataO,type);
    assert(hipGetLastError() == hipSuccess);
    std::swap(dataI,dataO);
  }
}

__global__ void polynomialNTTMul(cuyasheint_t *a,const cuyasheint_t *b,const int size){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      uint64_t a_value = a[tid];
      uint64_t b_value = b[tid];

      // In-place
      a[tid] = s_mul(a_value,b_value);
      // a[tid] = a_value*b_value % 18446744069414584321;
  }
}
#endif

__global__ void polynomialOPInteger(const int opcode,
                                      const cuyasheint_t *a,
                                      const cuyasheint_t *integer_array,
                                      cuyasheint_t *output,
                                      const int N,
                                      const int NPolis){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int size = N*NPolis;
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int cid = tid % N; // Coefficient id
  const int rid = tid / N; // Residue id

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.

    switch(opcode)
    {
    case ADD:
      if(cid == 0)
        output[tid] = a[tid] + integer_array[rid];
      break;
    case SUB:
      if(cid == 0)
        output[tid] = a[tid] - integer_array[rid];
      break;
    case DIV:
        output[tid] = a[tid] / integer_array[rid];
      break;
    case MUL:
        output[tid] = a[tid] * integer_array[rid];
      break;
    case MOD:
        output[tid] = a[tid] % integer_array[rid];
      break;
    default:
      //This case shouldn't be used. I will use 42 to recognize if we got in this case.
      output[tid] = 42;
      break;
    }
  }

}

__host__ cuyasheint_t* CUDAFunctions::callPolynomialOPInteger(const int opcode,
                                                      hipStream_t stream,
                                                      cuyasheint_t *a,
                                                      cuyasheint_t *integer_array,
                                                      const int N,
                                                      const int NPolis)
{
  // This method applies a 0-degree operation over all CRT residues
  const int size = N*NPolis;

  const int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  const dim3 gridDim(ADDGRIDXDIM);
  const dim3 blockDim(ADDBLOCKXDIM);

  cuyasheint_t *d_pointer;
  hipError_t result = hipMalloc((void**)&d_pointer,
                                  N*NPolis*sizeof(cuyasheint_t));        
  assert(result == hipSuccess);

  polynomialOPInteger<<< gridDim,blockDim, 1, stream>>> ( opcode,
                                                          a,
                                                          integer_array,
                                                          d_pointer,
                                                          N,
                                                          NPolis);
  assert(hipGetLastError() == hipSuccess);

  return d_pointer;
}

__host__ cuyasheint_t* CUDAFunctions::callPolynomialMul(hipStream_t stream,
                                                        cuyasheint_t *a,
                                                        const bool realign_A,
                                                        const int A_N,
                                                        cuyasheint_t *b,
                                                        const bool realign_B,
                                                        const int B_N,
                                                        const int N,
                                                        const int NPolis){
  // This method expects that both arrays are aligned

  // Input:
  // stream: cudaStream
  // a: first operand
  // realign_A: flag. true if this operand need to be realign
  // A_N: number of coefficients for each operand. Used only if we need to realign this
  // b: second operand
  // realign_B: flag. true if this operand need to be realign
  // B_N: number of coefficients for each residue. Used only if we need to realign this
  // N: number of coefficients for each residue. This is the N that should be considered for the operation.
  // NPolis: number of residues
  // All representations should be concatenated aligned
  assert((N>0)&&((N & (N - 1)) == 0));//Check if N is power of 2
  assert(N == CUDAFunctions::N);
  cuyasheint_t *d_result;

  #ifdef PLAINMUL
    // #ifdef VERBOSE
    //     std::cout << "Plain multiplication" << std::endl;
    // #endif
    // hipError_t result = hipMalloc((void**)&d_result,(N)*NPolis*sizeof(cuyasheint_t));
    // assert(result == hipSuccess);
    // result = hipMemset((void*)d_result,0,(N)*NPolis*sizeof(cuyasheint_t));
    // assert(result == hipSuccess);

    // dim3 blockDim(ADDBLOCKXDIM,ADDBLOCKXDIM);
    // // int blocks = ((2*N*NPolis) % ADDBLOCKXDIM == 0? (2*N*NPolis)/ADDBLOCKXDIM : (2*N*NPolis)/ADDBLOCKXDIM+1);
    // // dim3 gridDim(blocks,blocks);
    // dim3 gridDim(N*NPolis,1);
    // polynomialPlainMul<<<gridDim,blockDim>>>(a,b,d_result,N,NPolis);
    // assert(hipGetLastError() == hipSuccess);
  #elif defined(NTTMUL)

      // std::cout << "NTT multiplication" << std::endl;

  // Allocates memory for temporary arrays on device
  // Each polynomial's degree gets doubled
  //
  // d_result is used as auxiliar array
  const int size = N*NPolis;
  cuyasheint_t *d_a;
  cuyasheint_t *d_b;
  cuyasheint_t *aux;

  hipError_t result;
  result = hipMalloc((void**)&d_result,size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);
  result = hipMalloc((void**)&d_a,size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);
  result = hipMalloc((void**)&d_b,size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);
  result = hipMalloc((void**)&aux,size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);

  result = hipMemsetAsync(aux,0,size*sizeof(cuyasheint_t),stream);
  assert(result == hipSuccess);
  result = hipMemsetAsync(d_result,0,size*sizeof(cuyasheint_t),stream);
  assert(result == hipSuccess);

  result = hipMemcpyAsync(d_a,a,size*sizeof(cuyasheint_t),hipMemcpyDeviceToDevice,stream);
  assert(result == hipSuccess);
  result = hipMemcpyAsync(d_b,b,size*sizeof(cuyasheint_t),hipMemcpyDeviceToDevice,stream);
  assert(result == hipSuccess);

  const int RADIX = 2;
	dim3 blockDim(std::min(N/RADIX,1024));
	dim3 gridDim(NPolis);

  // Forward
  for(int Ns=1; Ns<N; Ns*=RADIX){
    NTT<<<gridDim,blockDim,1,stream >>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,RADIX,Ns,d_a,aux,FORWARD);
    assert(hipGetLastError() == hipSuccess);
    std::swap(aux,d_a);
  }
  

  result = hipMemsetAsync(aux,0,size*sizeof(cuyasheint_t),stream);
  assert(result == hipSuccess);

  for(int Ns=1; Ns<N; Ns*=RADIX){
    NTT<<<gridDim,blockDim,1,stream >>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,RADIX,Ns,d_b,aux,FORWARD);
    assert(hipGetLastError() == hipSuccess);
    std::swap(aux,d_b);
  }

  // Multiply
  dim3 blockDimMul(ADDBLOCKXDIM);
  dim3 gridDimMul((size)/ADDBLOCKXDIM+1); // We expect that ADDBLOCKXDIM always divide size
  polynomialNTTMul<<<gridDimMul,blockDimMul,1,stream>>>(d_a,d_b,size);
  assert(hipGetLastError() == hipSuccess);

  // // Inverse
  for(int Ns=1; Ns<N; Ns*=RADIX){
    NTT<<<gridDim,blockDim,1,stream >>>(CUDAFunctions::d_W,CUDAFunctions::d_WInv,N,RADIX,Ns,d_a,d_result,INVERSE);
    assert(hipGetLastError() == hipSuccess);
    std::swap(d_a,d_result);
  }

  std::swap(d_a,d_result);

  NTTScale<<< gridDimMul,blockDimMul,1,stream >>>(d_result,size,N);
  
  hipFree(d_a);
  hipFree(d_b);
  hipFree(aux);

  #elif defined(FFTMUL)

  // Allocates memory for temporary arrays on device
  // Each polynomial's degree gets doubled
  const int size = N*NPolis;
  hipError_t result = hipMalloc((void**)&d_result,size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);

  Complex *d_A;
  Complex *d_B;
  result = hipMalloc((void**)&d_A,size*sizeof(Complex));
  assert(result == hipSuccess);
  result = hipMalloc((void**)&d_B,size*sizeof(Complex));
  assert(result == hipSuccess);

  dim3 blockDim(ADDBLOCKXDIM);
  dim3 gridDim((size)/ADDBLOCKXDIM+1); // We expect that ADDBLOCKXDIM always divice size

  assert(blockDim.x*gridDim.x >= N);
  // Forward
  fft_radix16<<<gridDim,blockDim>>>(a,d_result,N);
  assert(hipGetLastError() == hipSuccess);

  // Multiply
  // polynomialFFTMul<<<gridDim,blockDim>>>(d_A,d_B,N*NPolis);

  // Inverse
  // fft_radix16<<<gridDim,blockDim>>>(d_A,d_result,N,NPolis,INVERSE);
  // assert(hipGetLastError() == hipSuccess);

  hipFree(d_A);
  hipFree(d_B);

  #elif defined(CUFFTMUL)
  
  const int size = N*NPolis;
  const int size_c = N;
  const int signal_size = N;
  Complex *d_a;
  Complex *d_b;
  Complex *d_c;
  hipError_t result;

  result = hipMalloc((void**)&d_result,size*sizeof(cuyasheint_t));
  assert(result == hipSuccess);

  result = hipMalloc((void **)&d_a, size*sizeof(Complex));
  assert(result == hipSuccess);

  result = hipMalloc((void **)&d_b, size*sizeof(Complex));
  assert(result == hipSuccess);

  result = hipMalloc((void **)&d_c, size*sizeof(Complex));
  assert(result == hipSuccess);
  
  result = hipMemsetAsync(d_a,0,size*sizeof(Complex),stream);
  assert(result == hipSuccess);
  result = hipMemsetAsync(d_b,0,size*sizeof(Complex),stream);
  assert(result == hipSuccess);
  result = hipMemsetAsync(d_c,0,size*sizeof(Complex),stream);
  assert(result == hipSuccess);

  dim3 blockDim(32);
  dim3 gridDim(size/32 + (size % 32 == 0? 0:1));

  if(realign_A){
    int size;
    const int newSpacing = N;
    const int oldSpacing = A_N;

    if(newSpacing < oldSpacing)
      size = newSpacing*NPolis;
    else
      size = oldSpacing*NPolis;

    const int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
    dim3 gridDim(ADDGRIDXDIM);
    dim3 blockDim(ADDBLOCKXDIM);
    copyAndRealignIntegerToComplex<<< gridDim,blockDim,1,stream >>>(d_a,a,A_N,N,NPolis);
  }else
    copyIntegerToComplex<<< gridDim,blockDim,1,stream >>>(d_a,a,size);
  assert(hipGetLastError() == hipSuccess);

  if(realign_B){
    int size;
    const int newSpacing = N;
    const int oldSpacing = B_N;

    if(newSpacing < oldSpacing)
      size = newSpacing*NPolis;
    else
      size = oldSpacing*NPolis;

    const int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
    dim3 gridDim(ADDGRIDXDIM);
    dim3 blockDim(ADDBLOCKXDIM);
    copyAndRealignIntegerToComplex<<< gridDim,blockDim,1,stream >>>(d_b,b,B_N,N,NPolis);
  }
  else
    copyIntegerToComplex<<< gridDim,blockDim,1,stream >>>(d_b,b,size);
  assert(hipGetLastError() == hipSuccess);

  

  // Stream set
  hipfftResult fftResult;
  fftResult = hipfftSetStream(CUDAFunctions::plan, stream);
  assert(fftResult == HIPFFT_SUCCESS);

  // Apply FFT
  fftResult = hipfftExecZ2Z(CUDAFunctions::plan, (hipfftDoubleComplex *)(d_a), (hipfftDoubleComplex *)(d_a), HIPFFT_FORWARD);
  assert(fftResult == HIPFFT_SUCCESS);

  fftResult = hipfftExecZ2Z(CUDAFunctions::plan, (hipfftDoubleComplex *)(d_b), (hipfftDoubleComplex *)(d_b), HIPFFT_FORWARD);
  assert(fftResult == HIPFFT_SUCCESS);

  polynomialcuFFTMul<<<gridDim,blockDim,1,stream>>>(d_a,d_b,d_c,size_c,size);
  assert(hipGetLastError() == hipSuccess);

  // Apply inverse FFT
  fftResult = hipfftExecZ2Z(CUDAFunctions::plan, (hipfftDoubleComplex *)(d_c), (hipfftDoubleComplex *)(d_c), HIPFFT_BACKWARD);
  assert(fftResult == HIPFFT_SUCCESS);

  copyAndNormalizeComplexRealPartToInteger<<< gridDim,blockDim,1,stream >>>(d_result,(hipfftDoubleComplex *)d_c,size,1.0f/signal_size,N);
  assert(hipGetLastError() == hipSuccess);


  //Destroy CUFFT context
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);


  #endif

  return d_result;
}

__host__ void CUDAFunctions::init(int N){
  CUDAFunctions::N = N;

  #ifdef NTTMUL
  #ifdef VERBOSE
  std::cout << "Will compute W -- N = " << N << std::endl;
  #endif

  cuyasheint_t *h_W;
  cuyasheint_t *h_WInv;

  assert((PZZ-1)%(N) == 0);

  cuyasheint_t k = conv<cuyasheint_t>(PZZ-1)/N;
  ZZ wNZZ = NTL::PowerMod(ZZ(PRIMITIVE_ROOT),k,PZZ);

  wN = conv<cuyasheint_t>(wNZZ);
  hipError_t result;
  h_W = (cuyasheint_t*)malloc(N*sizeof(cuyasheint_t));
  result = hipMalloc((void**)&d_W,N*sizeof(cuyasheint_t));
  assert(result == hipSuccess);
  h_WInv = (cuyasheint_t*)malloc(N*sizeof(cuyasheint_t));
  result = hipMalloc((void**)&d_WInv,N*sizeof(cuyasheint_t));
  assert(result == hipSuccess);

  // Computes 1-th column from W
  for(int j = 0; j < N; j++)
    h_W[j] = conv<cuyasheint_t>(NTL::PowerMod(wNZZ,j,PZZ));
  

  // Computes 1-th column from WInv
  for(int j = 0; j < N; j++)
      h_WInv[j] = conv<cuyasheint_t>(NTL::InvMod(conv<ZZ>(h_W[j]),PZZ ));

  result = hipMemcpy (d_W,h_W , N*sizeof(cuyasheint_t),hipMemcpyHostToDevice);
  assert(result == hipSuccess);
  result = hipMemcpy(d_WInv,h_WInv , N*sizeof(cuyasheint_t),hipMemcpyHostToDevice);
  assert(result == hipSuccess);

  free(h_W);
  free(h_WInv);
  #elif defined(CUFFTMUL)
    hipfftResult fftResult;

    // # of CRT residues
    const int batch = Polynomial::CRTPrimes.size();
    assert(batch > 0);

    // # 1 dimensional FFT
    const int rank = 1;

    // No idea what is this
    int n[1] = {N};


    fftResult = hipfftPlanMany(&CUDAFunctions::plan, rank, n,
         NULL, 1, N,  //advanced data layout, NULL shuts it off
         NULL, 1, N,  //advanced data layout, NULL shuts it off
         HIPFFT_Z2Z, batch);
    // fftResult = hipfftPlan1d(&CUDAFunctions::plan, N, HIPFFT_Z2Z, 1);


    assert(fftResult == HIPFFT_SUCCESS);
    std::cout << "Plan created with signal size " << N << std::endl;
    #ifdef VERBOSE
    #endif
  #endif
    /**
     * Alloc memory for d_inner_results
     */
    
    /**
     * For some reason
     */
    const unsigned int size = N*Polynomial::CRTPrimes.size();

    result = hipMalloc((void**)&CUDAFunctions::d_inner_results, size*STD_BNT_WORDS_ALLOC*sizeof(cuyasheint_t));
    assert(result == hipSuccess);
    result = hipMalloc((void**)&CUDAFunctions::d_inner_results_used, size*sizeof(cuyasheint_t));
    assert(result == hipSuccess);

}


// __global__ void polynomialReduction(cuyasheint_t *a,const const int half,const int N,const int NPolis){     
//   // This kernel must have (N-half)*Npolis threads

//   const int tid = threadIdx.x + blockIdx.x*blockDim.x;
//   const int residueID = tid / (N-half); 
//   const int cid = tid % (N-half);

//   if( (cid+half+1 < N) && (residueID*N + cid + half + 1 < N*NPolis)){
//     a[residueID*N + cid] = a[residueID*N + cid] - a[residueID*N + cid + half + 1];
//     __syncthreads();
//     a[residueID*N + cid + half + 1] = 0;
//   }
// }

// __host__ void Polynomial::reduce(){
//   // Just like DivRem, but here we reduce a with a cyclotomic polynomial
//   Polynomial *phi = Polynomial::global_phi;
//   ZZ q = Polynomial::global_mod;

//   if(!this->get_crt_computed()){
//     #ifdef VERBOSE
//     std::cout << "Reduce on host." << std::endl;
//     #endif
//     Polynomial quot;
//     Polynomial rem;
//     Polynomial::DivRem((*this),phi,quot, rem);
//     this->copy(rem);
//     this->update_crt_spacing(this->deg()+1);
//   }else{

//     modn(q);

//     #ifdef VERBOSE
//     std::cout << "Reduce on device." << std::endl;
//     #endif
    
//     const int half = phi->deg()-1;
//     const int N = this->get_crt_spacing();
//     const int NPolis = this->CRTPrimes.size();
//     const int size = (N-half)*NPolis;

//     if(size > 0){
//       dim3 blockDim(32);
//       dim3 gridDim(size/32 + (size % 32 == 0? 0:1));

//       polynomialReduction<<< gridDim,blockDim, 1, this->get_stream()>>>( this->get_device_crt_residues(),
//                                                                         half,
//                                                                         N,
//                                                                         NPolis);
//       hipError_t result = hipGetLastError();
//       assert(result == hipSuccess);
      
//       this->set_host_updated(false);
//       this->set_crt_residues_computed(true);
//       this->update_crt_spacing(phi->deg());
//     }
//   }
// }

// __global__ void cuGetHigher(int *result,const cuyasheint_t *a,const int N){
//   /**
//    * This kernel may be executed by N*NPolis/(32^iteration) threads
//    */
//   __shared__ int sdata[32];
//   int pdata = 0;

//   const unsigned int tid = threadIdx.x + blockIdx.x*blockDim.x;
//   if(tid < N){
//     sdata[threadIdx.x] = a[tid];
//     __syncthreads();

//   }

//   // Operation
//   while(i < n){
//     sdata[tid] = (a[tid] > a[tid+blockSize]?a[tid]:a[tid+blockSize]);
//     i += gridSize;
//   }

//  
// }

// __global__ void polynomialReduction(bn_t *a,const int half,const int N,const bn_t q){     
//   ////////////////////////////////////////////////////////
//   // This kernel must be executed with (N-half) threads //
//   ////////////////////////////////////////////////////////

//   const int tid = threadIdx.x + blockIdx.x*blockDim.x;
//   const int cid = tid % (N-half);

//   if(cid+half+1 < N){
//     const int flag_neg = bn_cmp_abs(&a[cid], &a[cid+half+1]);

//     assert(a[cid].alloc > a[cid+half+1].used);
//     a[cid].sign = bn_subn_low(a[cid].dp, a[cid].dp, a[cid+half+1].dp, a[cid+half+1].used);

//     __syncthreads();
//     bn_zero(&a[cid + half + 1]);
//   }
// }

__global__ void polynomialReduction(cuyasheint_t *a,const int half,const int N,const int NPolis){     
  // This kernel must have (N-half)*Npolis threads

  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  const int residueID = tid / (N-half); 
  const int cid = tid % (N-half);

  if( (cid+half+1 < N) && (residueID*N + cid + half + 1 < N*NPolis)){
    a[residueID*N + cid] -= a[residueID*N + cid + half + 1];
    __syncthreads();
    a[residueID*N + cid + half + 1] = 0;
  }

}

__host__ void Polynomial::reduce(){
  // Just like DivRem, but here we reduce with a cyclotomic polynomial
  
  //////////////////////////
  // Polynomial reduction //
  //////////////////////////

  Polynomial *phi = (Polynomial::global_phi);
  ZZ q = (Polynomial::global_mod);
  
  // Until we debug reduction on GPU, we need this
  update_host_data();
  this->set_crt_computed(false);

  if(!this->get_crt_computed()){
    #ifdef VERBOSE
    std::cout << "Reduce on host." << std::endl;
    #endif
    /**
     * Reduce on host
     */
    if(check_special_rem_format(phi)){
      #ifdef VERBOSE
      std::cout << "Rem in special mode."<<std::endl;
      #endif

      const unsigned int half = phi->deg()-1;     

      // #pragma omp parallel for
      for(unsigned int i = 0;i <= half;i++){
        this->set_coeff(i,this->get_coeff(i)-this->get_coeff(i+half+1));
        this->set_coeff(i+half+1,0);
      }
    }else{
      throw "Reduce: I don't know how to compute this!";
    }
    *this %= q;
    this->normalize();
    // this->update_crt_spacing(this->deg()+1);
  }else{

    #ifdef VERBOSE
    std::cout << "Reduce on device." << std::endl;
    #endif
    /**
     * Reduce on devicce
     */
    
    const int half = phi->deg()-1;
    const int N = this->get_crt_spacing();
    const int NPolis = this->CRTPrimes.size();
    const int size = (N-half)*NPolis;

    if(size > 0){
      dim3 blockDim(ADDBLOCKXDIM);
      dim3 gridDim(size/ADDBLOCKXDIM + (size % ADDBLOCKXDIM == 0? 0:1));

      polynomialReduction<<< gridDim,blockDim, 1, this->get_stream()>>>( this->get_device_crt_residues(),
                                                                          half,
                                                                          N,
                                                                          NPolis);
      hipError_t result = hipGetLastError();
      assert(result == hipSuccess);
      
      this->set_host_updated(false);
      this->set_crt_computed(true);
      this->set_icrt_computed(false);
      
      ///////////////////////
      // Modular reduction //
      ///////////////////////
      // icrt();
      // modn(q);
    }

  }
}
