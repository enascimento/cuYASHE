#include "hip/hip_runtime.h"
#include "cuda_functions.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>

///////////////////////////////////////
/// Memory operations
__global__ void realignCRTResidues(int oldSpacing,int newSpacing, long *array,long *new_array,int residuesSize,int residuesQty){
  //
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  const int residueId = tid / residuesSize;
  const int new_array_offset = (tid % residuesSize) + residueId*newSpacing;
  const int old_array_offset = (tid % residuesSize) + residueId*oldSpacing;

  if(new_array_offset < newSpacing*residuesQty && old_array_offset < oldSpacing*residuesQty )
    new_array[new_array_offset] = array[old_array_offset];

}


__host__ long* CUDAFunctions::callRealignCRTResidues(hipStream_t stream,int oldSpacing,int newSpacing, long *array,int residuesSize,int residuesQty){
  if(oldSpacing == newSpacing)
    return NULL;
  const int size = residuesSize*residuesQty;
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  long *d_new_array;
  hipError_t result = hipMalloc((void**)&d_new_array,newSpacing*residuesQty*sizeof(long));
  #ifdef VERBOSE
  std::cout << "hipMalloc:" << hipGetErrorString(result) << " "<< newSpacing*residuesQty*sizeof(long) << " bytes" <<std::endl;
  #endif
  assert(result == hipSuccess);

  realignCRTResidues <<< gridDim,blockDim,1,stream >>> (oldSpacing,newSpacing,array,d_new_array,residuesSize,residuesQty);
  #ifdef VERBOSE
  std::cout << gridDim.x << " " << blockDim.x << std::endl;
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif
  assert(hipGetLastError() == hipSuccess);

  return d_new_array;
}

///////////////////////////////////////

///////////////////////////////////////
/// ADD
__global__ void polynomialAddSub(const int OP,const long *a,const long *b,long *c,const int size){
  // We have one thread per polynomial coefficient on 32 threads-block.
  // For CRT polynomial adding, all representations should be concatenated aligned
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  long a_value;
  long b_value;

  if(tid < size ){
      // Coalesced access to global memory. Doing this way we reduce required bandwich.
      a_value = a[tid];
      b_value = b[tid];

      if(OP == ADD)
        a_value += b_value;
      else
        a_value -= b_value;

      c[tid] = a_value;
  }
}

__host__ long* CUDAFunctions::callPolynomialAddSub(hipStream_t stream,long *a,long *b,int size,int OP){
  // This method expects that both arrays are aligned
  int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? size/ADDBLOCKXDIM : size/ADDBLOCKXDIM + 1);
  dim3 gridDim(ADDGRIDXDIM);
  dim3 blockDim(ADDBLOCKXDIM);

  long *d_new_array;
  hipError_t result = hipMalloc((void**)&d_new_array,size*sizeof(long));
  #ifdef VERBOSE
  std::cout << "hipMalloc:" << hipGetErrorString(result) << " "<< size*sizeof(long) << " bytes" <<std::endl;
  #endif
  assert(result == hipSuccess);

  // polynomialAdd <<< gridDim,blockDim, 0, stream >>> (a,b,d_new_array,size);
  polynomialAddSub <<< gridDim,blockDim >>> (OP,a,b,d_new_array,size);
  #ifdef VERBOSE
  std::cout << gridDim.x << " " << blockDim.x << std::endl;
  std::cout << "polynomialAdd kernel:" << hipGetErrorString(hipGetLastError()) << std::endl;
  #endif

  return d_new_array;
}
///////////////////////////////////////

///////////////////////////////////////
/// MUL

__device__ void sumReduce(long value,long *a,int i,long q,int N, int NPolis){
  // Sum all elements in array "r" and writes to a, in position i

  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  __shared__ long r[ADDBLOCKXDIM];
  r[threadIdx.x] = value;

  if(tid < N*NPolis){

    int stage = blockDim.x;
    while(stage > 0){// Equivalent to for(int i = 0; i < lrint(log2(N))+1;i++)
      if(threadIdx.x < stage/2 && (tid % N) + stage/2 < N){
        // Only half of the threads are used
        r[threadIdx.x] += r[threadIdx.x + stage/2];
      }
      stage /= 2;
      __syncthreads();
    }
    // After this loop, r[0] hold the sum of all block data

    if(threadIdx.x == 0)
      atomicAdd((unsigned long long int*)(&(a[i])),(unsigned long long int)(r[threadIdx.x] % q));
    __syncthreads();
  }
}

__global__ void NTT(long *W,long *a, long *a_hat, long q, int N,int NPolis){
  // This algorithm supposes that N is power of 2, divisible by 32
  // Input:
  // w: Matrix of wNs
  // a: residues
  // a_hat: output
  // N: # of coefficients of each polynomial
  // NPolis: # of residues

  const int tid = threadIdx.x + blockIdx.x*blockDim.x;

  if(tid < N*NPolis){

    // In each iteration, computes a_hat[i]
    for(int i = 0; i < N; i++){
      int cid = tid % N; // Coefficient id

      sumReduce(W[cid + i*N]*a[cid],a_hat,i,q,N,NPolis);
    }
  }

}

__global__ void INTT(){
  // This algorithm supposes that N is power of 2

}


__host__ void host_NTT(dim3 gridDim,dim3 blockDim,long *W,long *a, long *a_hat, long q,int N,int NPolis){
  // This is a dummy method used by the test framework. Probably unnecessary.
  NTT<<<gridDim,blockDim>>>(W,a,a_hat,q,N,NPolis);
}


__host__ long* CUDAFunctions::callPolynomialMul(hipStream_t stream,long *a,long *b,int N,int NPolis){
  // All representations should be concatenated aligned
  assert((N>0)&&((N & (N - 1)) == 0));//Check if N is power of 2
  long *d_result;

  #ifdef PLAINMUL
    #ifdef VERBOSE
        std::cout << "Plain multiplication" << std::endl;
    #endif
  #else

    // To-do
    throw "Polynomial multiplication not implemented!";
    // Allocates memory for temporary arrays on device
    // Each polynomial's degree gets doubled
    long *d_a;
    long *d_b;
    hipError_t result = hipMalloc((void**)&d_a,2*size*sizeof(long));
    assert(result == hipSuccess);
    hipError_t result = hipMalloc((void**)&d_b,2*size*sizeof(long));
    assert(result == hipSuccess);

    dim3 blockDim(32);
    dim3 gridDim((2*size)/32+1);
    NTT<<<gridDim,blockDim,1,stream>>>(a,d_a,N,size);
  #endif

  return d_result;
}
