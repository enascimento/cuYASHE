#include <NTL/ZZ_pEX.h>

#include "polynomial.h"
#include "settings.h"
#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

int main(){
  int degree;
  Polynomial phi;

  degree = 8;

  Polynomial::global_mod = conv<ZZ>("655615111"); // Defines default GF(q)
  Polynomial::BuildNthCyclotomic(&phi,degree);
  phi.set_mod(Polynomial::global_mod);
  Polynomial::global_phi = &phi;

  srand (36251);

  Polynomial::gen_crt_primes(Polynomial::global_mod,degree);
  ZZ_p::init(Polynomial::global_mod);

  std::cout << "Phi: " << phi.to_string() << std::endl;

  // GPU
  Polynomial a;
  a.set_coeff(0,1);
  a.set_coeff(1,1);
  a.set_coeff(2,1);
  a.set_coeff(3,1);
  a.set_coeff(4,1);
  a.set_coeff(5,1);
  a.set_coeff(6,1);
  a.set_coeff(7,1);
  a.set_coeff(8,1);

  a.update_device_data();
  a.set_host_updated(false);
  a.reduce();

  std::cout << "GPU: " <<a.to_string() << std::endl;

  // CPU
  a.set_coeff(0,1);
  a.set_coeff(1,1);
  a.set_coeff(2,1);
  a.set_coeff(3,1);
  a.set_coeff(4,1);
  a.set_coeff(5,1);
  a.set_coeff(6,1);
  a.set_coeff(7,1);
  a.set_coeff(8,1);

  a.set_device_updated(false);
  a.set_host_updated(true);
  a.reduce();

  std::cout << "CPU: " <<a.to_string() << std::endl;


 // GPU
  a.set_coeff(0,722); 
  a.set_coeff(1,173); 
  a.set_coeff(2,735); 
  a.set_coeff(3,651); 
  a.set_coeff(4,460); 
  a.set_coeff(5,161); 
  a.set_coeff(6, 56); 
  a.set_coeff(7, 80); 
  a.set_coeff(8,762); 
  a.set_coeff(9,  9); 
  a.set_coeff(10,220); 
  a.set_coeff(11,281);  
  a.set_coeff(12, 62);  
  a.set_coeff(13, 96);  
  a.set_coeff(14,239);  

  a.update_device_data();
  a.set_host_updated(false);
  a.reduce();
  // a %= Polynomial::global_mod;
  
  std::cout << "GPU: " <<a.to_string() << std::endl;

  // CPU
  a.set_coeff(0,722); 
  a.set_coeff(1,173); 
  a.set_coeff(2,735); 
  a.set_coeff(3,651); 
  a.set_coeff(4,460); 
  a.set_coeff(5,161); 
  a.set_coeff(6, 56); 
  a.set_coeff(7, 80); 
  a.set_coeff(8,762); 
  a.set_coeff(9,  9); 
  a.set_coeff(10,220); 
  a.set_coeff(11,281);  
  a.set_coeff(12, 62);  
  a.set_coeff(13, 96);  
  a.set_coeff(14,239);  

  a.set_device_updated(false);
  a.set_host_updated(true);
  a.reduce();
  // a %= Polynomial::global_mod;
  
  std::cout << "CPU: " <<a.to_string() << std::endl;


}
