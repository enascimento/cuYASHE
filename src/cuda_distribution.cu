#include "settings.h"
#include "distribution.h"

__host__  void Distribution::callCuGetUniformSample(cuyasheint_t*array, int N){
	/**
	 * Generates N*NPolis random integers
	 */
	hiprandStatus_t result = hiprandGenerate( gen, 
											(unsigned int*)(array), 
											N);
	assert(result == HIPRAND_STATUS_SUCCESS);
}

__host__ void Distribution::callCuGetNormalSample(cuyasheint_t *array, int N, float mean, float stddev){
	hiprandGenerateLogNormal( gen, 
							(float*)array, 
							N,
							mean,
							stddev);

}