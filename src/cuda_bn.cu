#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include "cuda_bn.h"
#include "settings.h"
#include "cuda_functions.h"
#include "polynomial.h"

__constant__ cuyasheint_t CRTPrimesConstant[MAX_PRIMES_ON_C_MEMORY];

bn_t CUDAFunctions::M;
bn_t* CUDAFunctions::Mpis;
cuyasheint_t* CUDAFunctions::invMpis;

////////////////////////
// Auxiliar functions //
////////////////////////

__host__ __device__ int max_d(int a,int b){
	return (a >= b)*a + (b > a)*b;
}

__host__ __device__ int min_d(int a,int b){
	return (a <= b)*a + (b < a)*b;
}

__host__ __device__ void dv_zero(cuyasheint_t *a, int digits) {
	int i;
 
	// if (digits > DV_DIGS) {
	// 	std::cout << "ERR_NO_VALID" << std::endl;
	// 	exit(1);
	// }	
	for (i = 0; i < digits; i++, a++)
		(*a) = 0;

	return;
}

/**
 * Set a big number struct to zero
 * @param a operand
 */
__host__ __device__ void bn_zero(bn_t *a) {
	a->sign = BN_POS;
	a->used = 1;
	dv_zero(a->dp, a->alloc);
}

/**
 * Set a big number to digit
 * @param a     input: big number
 * @param digit input: digit
 */
__host__ __device__ void bn_set_dig(bn_t *a, cuyasheint_t digit) {
	bn_zero(a);	
	a->dp[0] = digit;
	a->used = 1;
	a->sign = BN_POS;
}

__host__ void bn_new(bn_t *a){
  a->used = 0;
  a->alloc = STD_BNT_ALLOC;
  a->sign = BN_POS;
  // std::cout << "Will alloc " << (a->alloc*sizeof(cuyasheint_t)) << " bytes" << std::endl;
  a->dp = (cuyasheint_t*) malloc(a->alloc*sizeof(cuyasheint_t));
}

// __device__ void bn_new_d(bn_t *a){
//   a->used = 0;
//   a->alloc = STD_BNT_ALLOC;
//   a->sign = BN_POS;
//   hipMalloc(&a->dp,a->alloc*sizeof(cuyasheint_t));
// }

__host__ void bn_free(bn_t *a){
  if(a->dp != NULL && a->alloc > 0){
	hipError_t result = hipFree((a->dp));
	if(result != hipSuccess){
		std::cout << hipGetErrorString(result) << std::endl;
		hipGetLastError();//Reset
	}
  	// assert(result == hipSuccess);
  }

  a->used = 0;
  a->alloc = 0;  

}

__host__ __device__ int bn_cmpn_low(const cuyasheint_t *a, const cuyasheint_t *b, int size) {
	int i, r;

	a += (size - 1);
	b += (size - 1);

	r = CMP_EQ;
	for (i = 0; i < size; i++, --a, --b) {
		if (*a != *b && r == CMP_EQ) {
			r = (*a > *b ? CMP_GT : CMP_LT);
		}
	}
	return r;
}


__host__ __device__ int bn_cmp_abs(const bn_t *a, const bn_t *b) {
	if (a->used > b->used) {
		return CMP_GT;
	}

	if (a->used < b->used) {
		return CMP_LT;
	}

	return bn_cmpn_low(a->dp, b->dp, a->used);
}

/**
 * Increase the allocated memory for a bn_t object.
 * @param a        input/output:operand
 * @param new_size input: new_size for dp
 */
__host__ void bn_grow(bn_t *a,const unsigned int new_size){
  // We expect that a->alloc <= new_size
  if((unsigned int)a->alloc > new_size)
  	return;

  std::cout << "Will alloc " << (new_size*sizeof(cuyasheint_t)) << " bytes" << std::endl;

  hipMalloc((void**)(&(a->dp)+a->alloc),new_size*sizeof(cuyasheint_t));
  a->alloc = new_size;

}

// __device__ void bn_grow_d(bn_t *a,const unsigned int new_size){
//   // We expect that a->alloc <= new_size
//   if((unsigned int)a->alloc >= new_size)
//   	return;

//   hipMalloc(&a->dp+a->alloc,new_size*sizeof(cuyasheint_t));
//   a->alloc = new_size;

// }

////////////////
// Operators //
//////////////

// Mod
__host__ __device__ cuyasheint_t bn_mod1_low(const cuyasheint_t *a,
									const int size,
									const cuyasheint_t b) {
	// Computes a % b
	
	dcuyasheint_t w;
	cuyasheint_t r;
	int i;

	w = 0;
	for (i = size - 1; i >= 0; i--) {
		w = (w << ((dcuyasheint_t)BN_DIGIT)) | ((dcuyasheint_t)a[i]);

		r = (cuyasheint_t)(w/b)*(w >= b);
		w -= (((dcuyasheint_t)r) * ((dcuyasheint_t)b))*(w >= b);
	}
	return (cuyasheint_t)w;
}

// Multiply 

/**
 * Computes a*digit
 * @param  c     output: result
 * @param  a     input: many-words first operand
 * @param  digit input: one-word second operand
 * @param  size  input: number of words in a
 * @return       output: result's last word
 */
__host__ __device__ cuyasheint_t bn_mul1_low(cuyasheint_t *c,
									const cuyasheint_t *a,
									cuyasheint_t digit,
									int size) {
	int i;
	cuyasheint_t carry;
	dcuyasheint_t r;

	carry = 0;
	for (i = 0; i < size; i++, a++, c++) {
		/* Multiply the digit *tmpa by b and accumulate with the previous
		 * result in the same columns and the propagated carry. */
		r = (dcuyasheint_t)(carry) + (dcuyasheint_t)(*a) * (dcuyasheint_t)(digit);
		/* Increment the column and assign the result. */
		*c = (cuyasheint_t)r;
		/* Update the carry. */
		carry = (cuyasheint_t)(r >> (dcuyasheint_t)BN_DIGIT);
	}

	return carry;
}

/**
 * Computes 64bits a*b mod c
 * @param result       output: result
 * @param a            input: first 64 bits operand
 * @param b            input: second 64 bits operand 
 * @param c 		   input: module
 */
__device__ void bn_64bits_mulmod(cuyasheint_t *result,
									cuyasheint_t a,
									cuyasheint_t b,
									cuyasheint_t m
									){
	/**
	 * http://stackoverflow.com/a/18680280/1541615
	 */
    uint64_t res = 0;
    uint64_t temp_b;

    /* Only needed if b may be >= m */
    if (b >= m) {
        if (m > UINT64_MAX / 2u)
            b -= m;
        else
            b %= m;
    }

    while (a != 0) {
        if (a & 1) {
            /* Add b to res, modulo m, without overflow */
            if (b >= m - res) /* Equiv to if (res + b >= m), without overflow */
                res -= m;
            res += b;
        }
        a >>= 1;

        /* Double b, modulo m */
        temp_b = b;
        if (b >= m - b)       /* Equiv to if (2 * b >= m), without overflow */
            temp_b -= m;
        b += temp_b;
    }
	*result = res;
}

// Add

/**
 * Computes a+b
 * @param  c    output: result
 * @param  a    input: many-words first operand
 * @param  b    input: many-words second operand
 * @param  size input: number of words to add
 * @return      output: result's last word
 */
__host__ __device__ cuyasheint_t bn_addn_low(cuyasheint_t *c,
									cuyasheint_t *a,
									cuyasheint_t *b,
									const int size
									) {
	int i;
	register cuyasheint_t carry, c0, c1, r0, r1;

	carry = 0;
	for (i = 0; i < size; i++, a++, b++, c++) {
		r0 = (*a) + (*b);
		c0 = (r0 < (*a));
		r1 = r0 + carry;
		c1 = (r1 < r0);
		carry = c0 | c1;
		(*c) = r1;
	}
	return carry;
}

/**
 * [bn_add1_low description]
 * @param  c     [description]
 * @param  a     [description]
 * @param  digit [description]
 * @param  size  [description]
 * @return       [description]
 */
__host__ __device__ cuyasheint_t bn_add1_low(cuyasheint_t *c, const cuyasheint_t *a, cuyasheint_t digit, int size) {
	int i;
	register cuyasheint_t carry, r0;

	carry = digit;
	for (i = 0; i < size && carry; i++, a++, c++) {
		r0 = (*a) + carry;
		carry = (r0 < carry);
		(*c) = r0;
	}
	for (; i < size; i++, a++, c++) {
		(*c) = (*a);
	}
	return carry;
}


////////////////////////
// Subtract
////////////////////////
/**
 * [bn_subn_low description]
 * @param  c    [description]
 * @param  a    [description]
 * @param  b    [description]
 * @param  size [description]
 * @return      [description]
 */
__host__ __device__ cuyasheint_t bn_subn_low(cuyasheint_t * c, const cuyasheint_t * a,
		const cuyasheint_t * b, int size) {
	int i;
	cuyasheint_t carry, r0, diff;

	/* Zero the carry. */
	carry = 0;
	for (i = 0; i < size; i++, a++, b++, c++) {
		diff = (*a) - (*b);
		r0 = diff - carry;
		carry = ((*a) < (*b)) || (carry && !diff);
		(*c) = r0;
	}
	return carry;
}

/**
 * Accumulates a double precision digit in a triple register variable.
 *
 * @param[in,out] R2		- most significant word of the triple register.
 * @param[in,out] R1		- middle word of the triple register.
 * @param[in,out] R0		- lowest significant word of the triple register.
 * @param[in] A				- the first digit to multiply.
 * @param[in] B				- the second digit to multiply.
 */
#define COMBA_STEP_BN_MUL_LOW(R2, R1, R0, A, B)								\
	dcuyasheint_t r = (dcuyasheint_t)(A) * (dcuyasheint_t)(B);										\
	cuyasheint_t _r = (R1);														\
	(R0) += (cuyasheint_t)(r);														\
	(R1) += (R0) < (cuyasheint_t)(r);												\
	(R2) += (R1) < _r;														\
	(R1) += (cuyasheint_t)((r) >> (dcuyasheint_t)BN_DIGIT);								\
	(R2) += (R1) < (cuyasheint_t)((r) >> (dcuyasheint_t)BN_DIGIT);

/**
 * Accumulates a single precision digit in a triple register variable.
 *
 * @param[in,out] R2		- most significant word of the triple register.
 * @param[in,out] R1		- middle word of the triple register.
 * @param[in,out] R0		- lowest significant word of the triple register.
 * @param[in] A				- the first digit to accumulate.
 */
#define COMBA_ADD(R2, R1, R0, A)											\
	cuyasheint_t __r = (R1);														\
	(R0) += (A);															\
	(R1) += (R0) < (A);														\
	(R2) += (R1) < __r;														\

__host__ __device__ void bn_muld_low(cuyasheint_t * c, const cuyasheint_t * a, int sa,
		const cuyasheint_t * b, int sb, int l, int h) {
	int i, j, ta;
	const cuyasheint_t *tmpa, *tmpb;
	cuyasheint_t r0, r1, r2;

	c += l;

	r0 = r1 = r2 = 0;
	for (i = l; i < sb; i++, c++) {
		tmpa = a;
		tmpb = b + i;
		for (j = 0; j <= i; j++, tmpa++, tmpb--) {
			COMBA_STEP_BN_MUL_LOW(r2, r1, r0, *tmpa, *tmpb);
		}
		*c = r0;
		r0 = r1;
		r1 = r2;
		r2 = 0;
	}
	ta = 0;
	for (i = sb; i < sa; i++, c++) {
		tmpa = a + ++ta;
		tmpb = b + (sb - 1);
		for (j = 0; j < sb; j++, tmpa++, tmpb--) {
			COMBA_STEP_BN_MUL_LOW(r2, r1, r0, *tmpa, *tmpb);
		}
		*c = r0;
		r0 = r1;
		r1 = r2;
		r2 = 0;
	}
	for (i = sa; i < h; i++, c++) {
		tmpa = a + ++ta;
		tmpb = b + (sb - 1);
		for (j = 0; j < sa - ta; j++, tmpa++, tmpb--) {
			COMBA_STEP_BN_MUL_LOW(r2, r1, r0, *tmpa, *tmpb);
		}
		*c = r0;
		r0 = r1;
		r1 = r2;
		r2 = 0;
	}
}


/**
 * [bn_mod_barrt description]
 * @param c  [description]
 * @param a  [description]
 * @param sa [description]
 * @param m  [description]
 * @param sm [description]
 * @param u  [description]
 * @param su [description]
 */

__device__ void bn_mod_barrt(bn_t *C, const bn_t *A,const int NCoefs,
		const cuyasheint_t * m,  int sm, const cuyasheint_t * u, int su) {

	/**
	 * Each thread handles one coefficient
	 */
	
	const int tid = threadIdx.x + blockDim.x*blockIdx.x;

	if(tid < NCoefs){
		cuyasheint_t *a = A[tid].dp;
		int sa = A[tid].used;
		cuyasheint_t *c = C[tid].dp;

		unsigned long mu;
		cuyasheint_t q[2*STD_BNT_ALLOC],t[2*STD_BNT_ALLOC],carry;
		int sq, st;
		int i;

		mu = sm;
		sq = sa - (mu - 1);
		for (i = 0; i < sq; i++) {
			q[i] = a[i + (mu - 1)];
		}

		if (sq > su) {
			bn_muld_low(t, q, sq, u, su, mu, sq + su);
		} else {
			bn_muld_low(t, u, su, q, sq, mu - (su - sq) - 1, sq + su);
		}
		st = sq + su;
		while (st > 0 && t[st - 1] == 0) {
			--(st);
		}

		sq = st - (mu + 1);
		for (i = 0; i < sq; i++) {
			q[i] = t[i + (mu + 1)];
		}

		if (sq > sm) {
			bn_muld_low(t, q, sq, m, sm, 0, sq + 1);
		} else {
			bn_muld_low(t, m, sm, q, sq, 0, mu + 1);
		}
		st = mu + 1;
		while (st > 0 && t[st - 1] == 0) {
			st--;
		}

		sq = mu + 1;
		for (i = 0; i < sq; i++) {
			q[i] = t[i];
		}

		st = mu + 1;
		for (i = 0; i < sq; i++) {
			t[i] = a[i];
		}
		carry = bn_subn_low(t, t, q, sq);
		while (st > 0 && t[st - 1] == 0) {
			st--;
		}

		if (carry) {
			sq = (mu + 1);
			for (i = 0; i < sq - 1; i++) {
				q[i] = 0;
			}
			q[sq - 1] = 1;
			bn_subn_low(t, q, t, sq);
		}

		while (bn_cmpn_low(t, m, sm) == 1) {
			bn_subn_low(t, t, m, sm);
		}

		for (i = 0; i < st; i++) {
			c[i] = t[i];
		}
	}
}


__global__ void cuModN(bn_t * c, const bn_t * a, const int NCoefs,
		const cuyasheint_t * m, int sm, const cuyasheint_t * u, int su){
	bn_mod_barrt(c,a,NCoefs,m,sm,u,su);
}

__host__ void callCuModN(bn_t * c, const bn_t * a,int NCoefs,
		const cuyasheint_t * m, int sm, const cuyasheint_t * u, int su,
		hipStream_t stream){

	const int size = NCoefs;
	int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? 
			size/ADDBLOCKXDIM : 
			size/ADDBLOCKXDIM + 1);
	dim3 gridDim(ADDGRIDXDIM);
	dim3 blockDim(ADDBLOCKXDIM);

	cuModN<<<gridDim,blockDim,0,stream>>>(c,a,NCoefs,m,sm,u,su);
}
/////////
// CRT //
/////////

/**
 * @d_polyCRT - output: array of residual polynomials
 * @x - input: array of coefficients
 * @ N - input: qty of coefficients
 * @NPolis - input: qty of primes/residual polynomials
 */
__global__ void cuCRT(	cuyasheint_t *d_polyCRT,
						const bn_t *x,
						const int used_coefs,
						const unsigned int N,
						const unsigned int NPolis
						){
	/**
	 * This function should be executed with used_coefs threads. 
	 * Each thread computes the residues for one coefficient
	 *
	 * x should be an array of N elements
	 * d_polyCRT should be an array of N*NPolis elements
	 */
	
	/**
	 * tid: thread id
	 * cid: coefficient id
	 * rid: residue id
	 */
	const unsigned int cid = threadIdx.x + blockIdx.x*blockDim.x;

	// x can be copied to shared memory!
	// 
	if(cid < used_coefs){
		for(unsigned int rid = 0; rid < NPolis; rid++)
			// Computes x mod pi
			d_polyCRT[cid + rid*N] = bn_mod1_low(	x[cid].dp,
													x[cid].used,
													CRTPrimesConstant[rid]
													);
	
	}
}	

/**
 * cuICRT computes ICRT on GPU
 * @param poly      output: An array of coefficients 
 * @param d_polyCRT input: The CRT residues
 * @param N         input: Number of coefficients
 * @param NPolis    input: Number of residues
 */
__global__ void cuICRT(	bn_t *poly,
						const cuyasheint_t *d_polyCRT,
						const unsigned int N,
						const unsigned int NPolis,
						const bn_t M,
						const bn_t *Mpis,
						const cuyasheint_t *invMpis,
						bn_t *inner_results
						){
	/**
	 * This function should be executed with N threads.
	 * Each thread j computes a Mpi*( invMpi*(value) % pi) and adds to poly[j]
	 */
	
	/**
	 * tid: thread id
	 * cid: coefficient id
	 * rid: residue id
	 */
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;
	const int cid = tid;
	
	 if(tid < N){

	 	if(poly[cid].alloc < 0)
	 		printf("Achei!\n");
	 	bn_zero(&poly[cid]);
 		
 		bn_t inner_result = inner_results[cid];
	 	for(unsigned int rid = 0; rid < NPolis;rid++){
				cuyasheint_t carry;
	 			cuyasheint_t x;

	 			// Get a prime
	 			cuyasheint_t pi = CRTPrimesConstant[rid];
	 	
	 			bn_zero(&inner_result);
	 	
	 			/**
	 			 * Inner
	 			 */
	 			bn_64bits_mulmod(	&x,
	 								invMpis[rid],
	 								d_polyCRT[cid + rid*N],
	 								pi);

	 			// Adjust available words in inner_result
 				assert(inner_result.alloc >= Mpis[rid].used+1);
 					// bn_grow_d(&inner_result,1);

	 			carry = bn_mul1_low(inner_result.dp,
		 					     	Mpis[rid].dp,
		 					     	x,
		 					     	Mpis[rid].used);
 				
 				inner_result.used = Mpis[rid].used;
	 			if(carry){
 					inner_result.dp[inner_result.used] = carry;	
	 				inner_result.used++;	 				
 				}

 				/**
 				 * Accumulate
 				 */

				bn_t a = ( bn_cmp_abs(&poly[cid],&inner_result) == CMP_GT? poly[cid] : inner_result );
				bn_t b = ( bn_cmp_abs(&poly[cid],&inner_result) == CMP_LT? poly[cid] : inner_result);

				int max = a.used;
				int min = b.used;

				/* Grow the result. */
				assert(poly[cid].alloc > max);

				if (a.used == b.used) {
					carry = bn_addn_low(poly[cid].dp, a.dp, b.dp, max);
				} else {
					carry = bn_addn_low(poly[cid].dp, a.dp, b.dp, min);
					carry = bn_add1_low(poly[cid].dp + min, a.dp + min, carry, max - min);
				}

				poly[cid].used = max;
				if (carry) {
					assert(poly[cid].alloc > max + 1);
					poly[cid].dp[max] = carry;
					poly[cid].used++;
				}

 				__syncthreads();
	 		}
 			bn_zero(&inner_result);

	 ////////////////////////////////////////////////
	 // To-do: Modular reduction of poly[cid] by M //
	 ////////////////////////////////////////////////
	 }

}

	/**
	 * This function should be executed with N*Npolis threads. 
	 * Each thread computes one coefficient of each residue of d_polyCRT
	 */
	
void callCRT(bn_t *coefs,const int used_coefs,cuyasheint_t *d_polyCRT,const int N, const int NPolis,hipStream_t stream){
	const int size = used_coefs;

	if(size <= 0)
		return;
	
	hipError_t result;

	int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? 
			size/ADDBLOCKXDIM : 
			size/ADDBLOCKXDIM + 1);
	dim3 gridDim(ADDGRIDXDIM);
	dim3 blockDim(ADDBLOCKXDIM);
	
	cuCRT<<<gridDim,blockDim,0,stream>>>(d_polyCRT,coefs,used_coefs,N,NPolis);
	result = hipGetLastError();
	assert(result == hipSuccess);
}
	/**
	 * This function should be executed with N threads.
	 * Each thread j computes a Mpi*( invMpi*(value) % pi) and adds to poly[j]
	 */

void callICRT(bn_t *coefs,cuyasheint_t *d_polyCRT,const int N, const int NPolis,hipStream_t stream){

	const int size = N;
	if(size <= 0)
		return;

	int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? 
			size/ADDBLOCKXDIM : 
			size/ADDBLOCKXDIM + 1);
	dim3 gridDim(ADDGRIDXDIM);
	dim3 blockDim(ADDBLOCKXDIM);

	cuICRT<<<gridDim,blockDim,0,stream>>>(	coefs,
											d_polyCRT,
											N,
											NPolis,
											CUDAFunctions::M,
											CUDAFunctions::Mpis,
											CUDAFunctions::invMpis,
											CUDAFunctions::d_inner_results);
	hipError_t result = hipGetLastError();
	assert(result == hipSuccess);
}

__host__ void  CUDAFunctions::write_crt_primes(){

  #ifdef VERBOSE
  std::cout << "primes: "<< std::endl;
  for(unsigned int i = 0; i < Polynomial::CRTPrimes.size();i++)
    std::cout << Polynomial::CRTPrimes[i] << " ";
  std::cout << std::endl;
  #endif
  
  // Choose what memory will be used to story CRT Primes
  if(Polynomial::CRTPrimes.size() < MAX_PRIMES_ON_C_MEMORY){
    
    #ifdef VERBOSE
    std::cout << "Writting CRT Primes to GPU's constant memory" << std::endl;
    #endif

    hipStream_t stream;
    hipStreamCreate(&stream);

    /////////////////
    // Copy primes //
    /////////////////
    hipError_t result = hipMemcpyToSymbolAsync(HIP_SYMBOL( CRTPrimesConstant),
                                              &(Polynomial::CRTPrimes[0]),
                                              Polynomial::CRTPrimes.size()*sizeof(cuyasheint_t),
                                              0,
	                                           hipMemcpyHostToDevice,
                                              stream
                                            );
    assert(result == hipSuccess);

    ////////////
    // Copy M //
    ////////////

    // if(M)
    	// hipFree(M);
    // hipMalloc((void**)&M,sizeof(bn_t));
    // get_words(M,Polynomial::CRTProduct);

    //////////////
    // Copy Mpi //
    //////////////
    

    bn_t *h_Mpis;
    h_Mpis = (bn_t*) malloc( Polynomial::CRTPrimes.size()*sizeof(bn_t) );

    if(CUDAFunctions::Mpis){
    	// Release
    	result = hipMemcpy(h_Mpis,CUDAFunctions::Mpis,Polynomial::CRTPrimes.size()*sizeof(bn_t),hipMemcpyDeviceToHost);
    	assert(result == hipSuccess);
    	for(unsigned int i = 0; i < Polynomial::CRTPrimes.size(); i++){
    		result = hipFree(h_Mpis[i].dp);
    		assert(result == hipSuccess);
    	}
    	hipFree(CUDAFunctions::Mpis);
    }

    result = hipMalloc((void**)&CUDAFunctions::Mpis,Polynomial::CRTPrimes.size()*sizeof(bn_t));
  	assert(result == hipSuccess);
    for(unsigned int i = 0; i < Polynomial::CRTPrimes.size();i++){
    	h_Mpis[i].alloc = 0;
    	get_words(&h_Mpis[i],Polynomial::CRTMpi[i]);
    }

	result = hipMemcpy(CUDAFunctions::Mpis,h_Mpis,Polynomial::CRTPrimes.size()*sizeof(bn_t),hipMemcpyHostToDevice);
	assert(result == hipSuccess);

    free(h_Mpis);
    /////////////////
    // Copy InvMpi //
    /////////////////

    if(CUDAFunctions::invMpis)
    	hipFree(CUDAFunctions::invMpis);
    result = hipMalloc((void**)&CUDAFunctions::invMpis,Polynomial::CRTPrimes.size()*sizeof(cuyasheint_t));
    assert(result == hipSuccess);

	result = hipMemcpyAsync(CUDAFunctions::invMpis,
							&Polynomial::CRTInvMpi[0],
							Polynomial::CRTPrimes.size()*sizeof(cuyasheint_t),
							hipMemcpyHostToDevice
							);
    assert(result == hipSuccess);


  }else{
    throw "Too many primes.";
  }
}