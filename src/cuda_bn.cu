#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <assert.h>
#include "cuda_bn.h"
#include "settings.h"
#include "cuda_functions.h"
#include "polynomial.h"

__constant__ cuyasheint_t CRTPrimesConstant[MAX_PRIMES_ON_C_MEMORY];

//////////////////////////
// CRT global variables //
//////////////////////////

__device__ bn_t* M;
__device__ bn_t* Mpis;
__device__ cuyasheint_t *invMpis;

////////////////////////
// Auxiliar functions //
////////////////////////


__host__ __device__ void dv_zero(cuyasheint_t *a, int digits) {
	int i;
 
	// if (digits > DV_DIGS) {
	// 	std::cout << "ERR_NO_VALID" << std::endl;
	// 	exit(1);
	// }
	hipError_t result = hipDeviceSynchronize();
	assert(result == hipSuccess);

	
	for (i = 0; i < digits; i++, a++)
		(*a) = 0;

	return;
}

/**
 * Set a big number struct to zero
 * @param a operand
 */
__host__ __device__ void bn_zero(bn_t *a) {
	a->sign = BN_POS;
	a->used = 1;
	dv_zero(a->dp, a->alloc);
}

/**
 * Set a big number to digit
 * @param a     input: big number
 * @param digit input: digit
 */
__host__ __device__ void bn_set_dig(bn_t *a, cuyasheint_t digit) {
	hipError_t result = hipDeviceSynchronize();
	assert(result == hipSuccess);

	bn_zero(a);	
	a->dp[0] = digit;
	a->used = 1;
	a->sign = BN_POS;
}

__host__ void bn_new(bn_t *a){
  a->used = 0;
  a->alloc = STD_BNT_ALLOC;
  a->sign = BN_POS;
  hipMallocManaged(&a->dp,a->alloc*sizeof(cuyasheint_t));
}

__host__ void bn_free(bn_t *a){
  a->used = 0;
  a->alloc = 0;
  
  hipError_t result = hipDeviceSynchronize();
  assert(result == hipSuccess);
  result = hipFree(a->dp);
  assert(result == hipSuccess);

}

/**
 * Increase the allocated memory for a bn_t object.
 * @param a        input/output:operand
 * @param new_size input: new_size for dp
 */
__host__ void bn_grow(bn_t *a,const unsigned int new_size){
  // We expect that a->alloc <= new_size
  if((unsigned int)a->alloc <= new_size)
  	return;

  hipMallocManaged(&a->dp+a->alloc,new_size*sizeof(cuyasheint_t));
  a->alloc = new_size;

}


////////////////
// Operators //
//////////////

// Mod
__device__ cuyasheint_t bn_mod1_low(const cuyasheint_t *a,
									const int size,
									const cuyasheint_t b) {
	// Computes a % b
	
	dcuyasheint_t w;
	cuyasheint_t r;
	int i;

	w = 0;
	for (i = size - 1; i >= 0; i--) {
		w = (w << ((dcuyasheint_t)BN_DIGIT)) | ((dcuyasheint_t)a[i]);

		if (w >= b) {
			r = (cuyasheint_t)(w / b);
			w -= ((dcuyasheint_t)r) * ((dcuyasheint_t)b);
		} else {
			r = 0;
		}
	}
	return (cuyasheint_t)w;
}

// Multiply 

/**
 * Computes a*digit
 * @param  c     output: result
 * @param  a     input: many-words first operand
 * @param  digit input: one-word second operand
 * @param  size  input: number of words in a
 * @return       output: result's last word
 */
__device__ cuyasheint_t bn_mul1_low(cuyasheint_t *c,
									const cuyasheint_t *a,
									cuyasheint_t digit,
									int size) {
	int i;
	cuyasheint_t carry;
	dcuyasheint_t r;

	carry = 0;
	for (i = 0; i < size; i++, a++, c++) {
		/* Multiply the digit *tmpa by b and accumulate with the previous
		 * result in the same columns and the propagated carry. */
		r = (dcuyasheint_t)(carry) + (dcuyasheint_t)(*a) * (dcuyasheint_t)(digit);
		/* Increment the column and assign the result. */
		*c = (cuyasheint_t)r;
		/* Update the carry. */
		carry = (cuyasheint_t)(r >> (dcuyasheint_t)BN_DIGIT);
	}
	
	// Adds carry as last word
	c++;
	*c = (cuyasheint_t)carry;

	return carry;
}

/**
 * Computes 64bits a*b mod c
 * @param result       output: result
 * @param a            input: first 64 bits operand
 * @param b            input: second 64 bits operand 
 * @param c 		   input: module
 */
__device__ void bn_64bits_mulmod(cuyasheint_t *result,
									cuyasheint_t a,
									cuyasheint_t b,
									cuyasheint_t c
									){

	uint64_t w;
	uint64_t r;
	const int size = 2;
	
	/**
	 * Mul
	 */
	const uint64_t A[2] = {a*b,__umul64hi(a,b)};

	/**
	 * Modular reduction
	 */
	w = 0;
	for (int i = size - 1; i >= 0; i--){
		w = (w << ((uint64_t)32)) | ((uint64_t)A[i]);

		if (w >= b) {
			r = (uint32_t)(w / b);
			w -= ((uint64_t)r) * ((uint64_t)b);
		} else {
			r = 0;
		}
	}

	*result = r;
}

// Add

/**
 * Computes a+b
 * @param  c    output: result
 * @param  a    input: many-words first operand
 * @param  b    input: many-words second operand
 * @param  size input: number of words to add
 * @return      output: result's last word
 */
__device__ cuyasheint_t bn_addn_low(cuyasheint_t *c,
									const cuyasheint_t *a,
									const cuyasheint_t *b,
									int size
									) {
	int i;
	register cuyasheint_t carry, c0, c1, r0, r1;

	carry = 0;
	for (i = 0; i < size; i++, a++, b++, c++) {
		r0 = (*a) + (*b);
		c0 = (r0 < (*a));
		r1 = r0 + carry;
		c1 = (r1 < r0);
		carry = c0 | c1;
		(*c) = r1;
	}
	return carry;
}

/////////
// CRT //
/////////

/**
 * @d_polyCRT - output: array of residual polynomials
 * @x - input: array of coefficients
 * @ N - input: qty of coefficients
 * @NPolis - input: qty of primes/residual polynomials
 */
__global__ void cuCRT(	cuyasheint_t *d_polyCRT,
						const bn_t *x,
						const int unsigned N,
						const unsigned int NPolis
						){
	/**
	 * This function should be executed with N*Npolis threads. 
	 * Each thread computes one coefficient of each residue of d_polyCRT
	 */
	
	/**
	 * tid: thread id
	 * cid: coefficient id
	 */
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;
	const int cid = tid & (N -1 ); // We expect that N is a power of two

	if(tid < N*NPolis){

		// pid == tid <=> prime's id
		// Load this thread's coefficient
		// Computes x mod pi
		d_polyCRT[cid + tid*N] = bn_mod1_low(	x[cid].dp,
												x[cid].used,
												CRTPrimesConstant[tid]
												);
	}
}	

/**
 * cuICRT computes ICRT on GPU
 * @param poly      output: An array of coefficients 
 * @param d_polyCRT input: The CRT residues
 * @param N         input: Number of coefficients
 * @param NPolis    input: Number of residues
 */
__global__ void cuICRT(	bn_t *poly,
						const cuyasheint_t *d_polyCRT,
						const int unsigned N,
						const unsigned int NPolis,
						cuyasheint_t *CRTPrimes
						){
	/**
	 * This function should be executed with N threads.
	 * Each thread j computes a Mpi*( invMpi*(value) % pi) and adds to poly[j]
	 */
	
	/**
	 * tid: thread id
	 * cid: coefficient id
	 * rid: residue id
	 */
	const int tid = threadIdx.x + blockIdx.x*blockDim.x;
	const int cid = tid & (N -1 ); // We expect that N is a power of two

	 if(tid < N){

	 	for(unsigned int rid = 0; rid < NPolis;rid++){
	 			// Get a prime
	 			cuyasheint_t pi = CRTPrimes[rid];
	 	
	 			// Computes the inner result
	 			bn_t inner_result;
	 			bn_zero(&inner_result);
	 			cuyasheint_t x;
	 	
	 			bn_64bits_mulmod(	&x,
	 								invMpis[rid],
	 								d_polyCRT[tid],
	 								pi);
	 			bn_mul1_low(	inner_result.dp,
	 					     	Mpis[rid].dp,
	 					     	x,
	 					     	Mpis[rid].used);
	 			
	 			bn_addn_low(poly[cid].dp,
							poly[cid].dp,
							inner_result.dp,
							inner_result.used
							);
	 		}

	 ////////////////////////////////////////////////
	 // To-do: Modular reduction of poly[cid] by M //
	 ////////////////////////////////////////////////
	 }

}


// __global__ void cuCRT(	cuyasheint_t *d_polyCRT,
// 						const bn_t *x,
// 						const int unsigned N,
// 						const unsigned int NPolis
// 						){
	/**
	 * This function should be executed with N*Npolis threads. 
	 * Each thread computes one coefficient of each residue of d_polyCRT
	 */
	
void crt(bn_t *coefs,cuyasheint_t *d_polyCRT,const int N, const int NPolis,hipStream_t stream){

	const int size = N*NPolis;
	int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? 
			size/ADDBLOCKXDIM : 
			size/ADDBLOCKXDIM + 1);
	dim3 gridDim(ADDGRIDXDIM);
	dim3 blockDim(ADDBLOCKXDIM);
	
	cuCRT<<<gridDim,blockDim,1,stream>>>(d_polyCRT,coefs,N,NPolis);
}

// __global__ void cuICRT(	bn_t *poly,
// 						const cuyasheint_t *d_polyCRT,
// 						const int unsigned N,
// 						const unsigned int NPolis
// 						){
	/**
	 * This function should be executed with N threads.
	 * Each thread j computes a Mpi*( invMpi*(value) % pi) and adds to poly[j]
	 */

void icrt(bn_t *coefs,cuyasheint_t *d_polyCRT,const int N, const int NPolis,hipStream_t stream){

	const int size = N;
	int ADDGRIDXDIM = (size%ADDBLOCKXDIM == 0? 
			size/ADDBLOCKXDIM : 
			size/ADDBLOCKXDIM + 1);
	dim3 gridDim(ADDGRIDXDIM);
	dim3 blockDim(ADDBLOCKXDIM);
	///////////
	// To-do //
	///////////
}

__host__ void  CUDAFunctions::write_crt_primes(){

  #ifdef VERBOSE
  std::cout << "primes: "<< std::endl;
  for(unsigned int i = 0; i < Polynomial::CRTPrimes.size();i++)
    std::cout << Polynomial::CRTPrimes[i] << " ";
  std::cout << std::endl;
  #endif
  
  // Choose what memory will be used to story CRT Primes
  if(Polynomial::CRTPrimes.size() < MAX_PRIMES_ON_C_MEMORY){
    
    #ifdef VERBOSE
    std::cout << "Writting CRT Primes to GPU's constant memory" << std::endl;
    #endif

    hipError_t result = hipMemcpyToSymbol(HIP_SYMBOL( CRTPrimesConstant),
                                              &(Polynomial::CRTPrimes[0]),
                                              Polynomial::CRTPrimes.size()*sizeof(cuyasheint_t)
                                            );
    assert(result == hipSuccess);
  }else{
    throw "Too many primes.";
  }
}