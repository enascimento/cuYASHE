#include <NTL/ZZ_pEX.h>

#include "polynomial.h"
#include "settings.h"
#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

int main(){
   int degree;
  Polynomial phi;
  ZZ_pX NTL_Phi;

  degree = 8;

  Polynomial::global_mod = conv<ZZ>("1171313591017775093490277364417"); // Defines default GF(q)
  Polynomial::BuildNthCyclotomic(&phi,degree);
  phi.set_mod(Polynomial::global_mod);
  Polynomial::global_phi = &phi;

  srand (36251);

  Polynomial::gen_crt_primes(Polynomial::global_mod,degree);
  
  ZZ_p::init(Polynomial::global_mod);
  for(int i = 0; i <= phi.deg();i++){
    NTL::SetCoeff(NTL_Phi,i,conv<ZZ_p>(phi.get_coeff(i)));
  }
  ZZ_pE::init(NTL_Phi);

  Polynomial a,b;
  Polynomial::random(&a,8);
  Polynomial::random(&b,8);

  std::cout << "a: " << a.to_string() << std::endl;
  std::cout << "b: " << b.to_string() << std::endl;

  Polynomial c = a*b;
  
  std::cout << "a: " << a.to_string() << std::endl;
  std::cout << "b: " << b.to_string() << std::endl;
  std::cout << "a*b: " <<c.to_string() << std::endl;
}
