#include <NTL/ZZ_pEX.h>

#include "polynomial.h"
#include "common.h"
#include "cuda_functions.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

int main(){
  int degree;
  Polynomial phi;

  degree = 32;

  Polynomial::global_mod = conv<ZZ>("61"); // Defines default GF(q)
  Polynomial::BuildNthCyclotomic(&phi,degree);
  // std::cout << phi.to_string() << std::endl;
  phi.set_mod(Polynomial::global_mod);
  Polynomial::global_phi = &phi;

  srand (36251);

  ZZ_p::init(Polynomial::global_mod);
  ZZ_pX NTL_Phi;
  for(int i = 0; i <= phi.deg();i++){
    NTL::SetCoeff(NTL_Phi,i,conv<ZZ_p>(phi.get_coeff(i)));
  }
  ZZ_pE::init(NTL_Phi);

  Polynomial::gen_crt_primes(Polynomial::global_mod,degree);


  Polynomial a,b;
  a.set_device_updated(false);
  b.set_device_updated(false);
  Polynomial::random(&a,degree-1);
  Polynomial::random(&b,degree-1);
  a.set_host_updated(true);
  b.set_host_updated(true);

  ZZ_pEX b_ntl;
  ZZ_pEX a_ntl;
  for(int i = 0;i <= a.deg();i++)
    NTL::SetCoeff(a_ntl,i,conv<ZZ_p>(a.get_coeff(i)));
  for(int i = 0;i <= b.deg();i++)
    NTL::SetCoeff(b_ntl,i,conv<ZZ_p>(b.get_coeff(i)));

  Polynomial c = a*b;
  c.icrt();

  ZZ_pEX c_ntl = a_ntl*b_ntl;

  std::cout << "c: " << c.to_string() << " degree: " << c.deg() << std::endl << std::endl;
}
